
/***************************************************************************
 *   Copyright (C) 2020                                                    *
 *                                                                         *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ***************************************************************************/


/**
	@author Broux Paul-Emmanuel <paulemmanuelb@gmail.com>
 */

#include "encryptKernelECB.h"
#include "decryptKernelECB.h"
#include "utils.h"
#include "sboxE.h"
#include "sboxD.h"

int main(int argc, char * argv[]) {


    ///////////////////////////////////////////////////////////////
    // command line arguments
    ///////////////////////////////////////////////////////////////
    int     warm_up_device      = 0;    // GPU kernel warm up
	int     warm_up_host        = 1;    // Host warm up (with GPU kernel inside)
    int     threadNum           = 512;  // Threads per block. This is a recommanded number.
    int     blockNum            = 0;    // Number of blocks in the grid
    int     mode                = 1;    // Encryption mode, 1 to encrypt or 0 to decrypt.
    char *  filename;
    char *  inputKey; 

    std::cout << std::endl << "********************************************************" << std::endl;
    std::cout << std::endl << "******                     AES CUDA               ******" << std::endl;
    std::cout << std::endl << "********************************************************" << std::endl;


    if (argc > 1){
        for( int n=1 ; n<argc ; n=n+2 ) {
            if((strcmp(argv[n],"-wuDevice") == 0) && (n+1<argc)) {
                warm_up_device = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-wuHost") == 0) && (n+1<argc)) {
                warm_up_host = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-threadNum") == 0) && (n+1<argc)) {
                threadNum = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-blockNum") == 0) && (n+1<argc)) {
                blockNum = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-mode") == 0) && (n+1<argc)) {
                mode = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-filename") == 0) && (n+1<argc)) {
                strcpy(filename,argv[n+1]);
            }
            else if((strcmp(argv[n],"-inputKey") == 0) && (n+1<argc)) {
                strcpy(inputKey,argv[n+1]);
            }
            else if((strcmp(argv[n],"-help") == 0)) {
                std::cout << "   This is a AES-128 implementation." << std::endl;
				std::cout << "   \"-options value\" availables are:" << std::endl;
                std::cout << "   -mode, 1 to encrypt and 0 to decrypt. Default value is 1." << std::endl;
                std::cout << "   -filename, the file path to encrypt or decrypt." << std::endl;
                std::cout << "   -inputKey, the 128 key file's path to use for encyption or decryption." << std::endl;
                std::cout << "   -threadNum to set the number of threads per block. Default recommended value is 512." << std::endl;
                std::cout << "   -blockNum to set the number of blocks in the grid. Default value is 0 and will create enough blocks
                taking into account the input file size and the threadNum argument." << std::endl;
                std::cout << "   -wuDevice number of device kernel launch before timing (around 1000 or less is adviced) " << std::endl;
                std::cout << "   -wuHost number of main Host code launch for several timing results (around 10 should be enough)" << std::endl;
                std::cout << "   The order between options is not important." << std::endl << std::endl;
                exit(0);
            }
            else {
                std::cout << "Argument " << argv[n] << " does not correspond to any valid arguments. Type -help for details
                about valid command line arguments." <<std::endl;
		        exit(0);
            }
        }


    }
	else {
		std::cout << "   Not enough arguments. Type -help option in the command line for further explanations." << std::endl;
		exit(0);
	}

    std::cout << "    mode		    = " << mode             << std::endl;
    std::cout << "    threadNum		= " << threadNum        << std::endl;
    std::cout << "    blockNum		= " << blockNum         << std::endl;
    std::cout << "    wuHost		= " << warm_up_host     << std::endl;
    std::cout << "    wuDevice		= " << warm_up_device   << std::endl;


    //Copying the key file
    const unsigned char * keyCryptFile;
    ifstream keyFile(filename, ios::binary);
    if(!keyFile) {
        cerr << "Read error";
        exit(1);
    }
    keyFile >> keyCryptFile;

    //Key scheduling
    AES_KEY *key;

    if(mode) {
        AES_cuda_set_encrypt_key(keyCryptFile, 128, key);
    }
    else {
        AES_cuda_set_decrypt_key(keyCryptFile, 128, key);
    }

    //Checking for the size of the file to encrypt/decrypt
    size_t filesize;
    filesize = filesize(filename);
    
    //CMS padding to have 16 bytes blocks of data
    uint8_t padElmt;
    uint8_t mod16 = filesize%16;

    padElmt = 16 - mod16; // We always add bytes for future padding detection

    uint32_t inputData[filesize+padElmt];
    uint32_t outputData[filesize];

    ifstream inputFile(filename, ios::binary);
    if(!inputFile) {
        cerr << "Read error";
        exit(1);
    }
    inputFile >> inputData;

    for (int i = 0; i < padElmt; i++) {
				inputData[filesize + i] = padElmt;
    }
	filesize += padElmt;

    
    //Determining grid size if not given
    if(!blockNum) {
        blockNum = 1+filesize/(threadNum*16);
    }

    //Device vectors declarations and allocations
    uint32_t * devIntput, * devOutput, * dev_sm_te1, * dev_sm_te2, * dev_sm_te3, * dev_sm_te4;
    hipMalloc(&devInput, filesize*sizeof(uint32_t));
    hipMalloc(&devOutput, filesize*sizeof(uint32_t));
    hipMalloc(&dev_sm_te1, 256*sizeof(uint32_t));
    hipMalloc(&dev_sm_te2, 256*sizeof(uint32_t));
    hipMalloc(&dev_sm_te3, 256*sizeof(uint32_t));
    hipMalloc(&dev_sm_te4, 256*sizeof(uint32_t));

    //Copy vectors from host memory to device memory
    if(mode) {
        hipMemcpy(dev_sm_te1, TBox0, 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te2, TBox1, 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te3, TBox2, 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te4, TBox3, 256*sizeof(uint32_t), hipMemcpyHostToDevice);
    }
    else {
        hipMemcpy(dev_sm_te1, TBoxi0, 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te2, TBoxi1, 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te3, TBoxi2, 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te4, TBoxi3, 256*sizeof(uint32_t), hipMemcpyHostToDevice);
    }
    hipMemcpy(devInput  , inputData, filesize*sizeof(uint32_t), hipMemcpyHostToDevice);
    

    if(mode) {
        for(int i=0; i < warm_up_device ; i++) {						
            encrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize, dev_sm_te1,
            dev_sm_te2, dev_sm_te3, dev_sm_te4);
	    }
    }
    else {
        for(int i=0; i < warm_up_device ; i++) {						
            decrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize, dev_sm_te1,
            dev_sm_te2, dev_sm_te3, dev_sm_te4);
	    }
    }

    //Copy results from host memory to device memory
	hipMemcpy(outputData, devOutput, (filesize-padElmt)*sizeof(uint32_t), hipMemcpyDeviceToHost);


    //
    ifstream outputFile("res", ios::binary);
    if(!outputFile) {
        cerr << "Read error";
        exit(1);
    }
    outputData >> outputFile;

    // Free device memory
    hipFree(devInput);
    hipFree(devOutput);
    hipFree(dev_sm_te1);
    hipFree(dev_sm_te2);
    hipFree(dev_sm_te3);
    hipFree(dev_sm_te4);
    


	return 0;

}
