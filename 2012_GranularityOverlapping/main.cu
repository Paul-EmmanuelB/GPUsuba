
/***************************************************************************
 *   Copyright (C) 2020                                                    *
 *                                                                         *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ***************************************************************************/


/**
	@author Broux Paul-Emmanuel <paulemmanuelb@gmail.com>
 */
#include <stdio.h>
#include <cstdlib>

#include "utils.h"
#include "typedef.h"
#include "encryptKernelECB.h"
#include "decryptKernelECB.h"
#include "sboxE.h"
#include "sboxD.h"


int main(int argc, char * argv[]) {


    ///////////////////////////////////////////////////////////////
    // command line arguments
    ///////////////////////////////////////////////////////////////
    int     warm_up_device      = 1;    // GPU kernel warm up
	int     warm_up_host        = 0;    // Host warm up (with GPU kernel inside)
    int     threadNum           = 512;  // Threads per block. This is a recommanded number.
    int     blockNum            = 0;    // Number of blocks in the grid
    int     mode                = 1;    // Encryption mode, 1 to encrypt or 0 to decrypt.
    char *  filename;
    char *  keyfilename; 

    std::cout << std::endl << "********************************************************************" ;
    std::cout << std::endl << "******                   AES-128 CUDA                         ******" ;
    std::cout << std::endl << "******                Updated project of                      ******" ;
    std::cout << std::endl << "******  Implementation and Analysis of AES Encryption on GPU  ******" ;
    std::cout << std::endl << "******                      FROM                              ******" ;
    std::cout << std::endl << "******                 Qinjian Li and Al                      ******" ;
    std::cout << std::endl << "********************************************************************" << std::endl << std::endl; 

    if (argc > 1){
        for( int n=1 ; n<argc ; n=n+2 ) {
            if((strcmp(argv[n],"-wuDevice") == 0) && (n+1<argc)) {
                warm_up_device = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-wuHost") == 0) && (n+1<argc)) {
                warm_up_host = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-threadNum") == 0) && (n+1<argc)) {
                threadNum = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-blockNum") == 0) && (n+1<argc)) {
                blockNum = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-mode") == 0) && (n+1<argc)) {
                mode = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-filename") == 0) && (n+1<argc)) {
                filename = argv[n+1];
            }
            else if((strcmp(argv[n],"-keyfilename") == 0) && (n+1<argc)) {
                keyfilename = argv[n+1];
            }
            else if((strcmp(argv[n],"-help") == 0)) {
                std::cout << "   This is a AES-128 implementation." << std::endl;
				std::cout << "   \"-options value\" availables are:" << std::endl;
                std::cout << "   -mode, 1 to encrypt and 0 to decrypt. Default value is 1." << std::endl;
                std::cout << "   -filename, the file path to encrypt or decrypt." << std::endl;
                std::cout << "   -keyfilename, the 128 key file's path to use for encyption or decryption." << std::endl;
                std::cout << "   -threadNum to set the number of threads per block. Default recommended value is 512." << std::endl;
                std::cout << "   -blockNum to set the number of blocks in the grid. Default value is 0 and will create enough blocks taking into account the input file size and the threadNum argument." << std::endl;
                std::cout << "   -wuDevice number of device kernel launch before timing (around 1000 or less is adviced) " << std::endl;
                std::cout << "   -wuHost number of main Host code launch for several timing results (around 10 should be enough)" << std::endl;
                std::cout << "   The order between options is not important." << std::endl << std::endl;
                exit(0);
            }
            else {
                std::cout << std::endl << "Argument " << argv[n] << " does not correspond to any valid arguments. Type -help for details about valid command line arguments." <<std::endl;
		        exit(1);
            }
        }


    }
	else {
		std::cout << std::endl << std::endl << "Not enough arguments. Type -help option in the command line for further explanations." << std::endl;
		exit(1);
	}

    std::cout << "    mode		 = " 	<< mode             << std::endl;
    std::cout << "    threadNum		= " << threadNum        << std::endl;
    std::cout << "    blockNum		= " << blockNum         << std::endl;
    std::cout << "    wuHost		= " << warm_up_host     << std::endl;
    std::cout << "    wuDevice		= " << warm_up_device   << std::endl;

    
    //Copying the key file
    unsigned char key[16];
    FILE * keyFile;
    keyFile = fopen(keyfilename,"rb");
    if (keyFile == NULL) {
        perror ("Error opening file");
        exit(1);
    }
    else {
        for(int i=0 ; i<16 ; i+=4) {
            if(fscanf(keyFile, "%x", (unsigned int *)&key[i]) != 1 ) {
                perror ("Error reading keyfile. Make sure the key is hexadecimal words like \"0x01234567 0x89abcdef ...\" .\n");
                exit(1);
            }
        }
    }
    fclose(keyFile);


    // ***Key scheduling***
    uint8 expkey[176];
    ExpandKey (key, expkey);
    hipMemcpyToSymbol(HIP_SYMBOL(const_expkey),  expkey, 176*sizeof(uint8)); //Moving the expanding key to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(const_IK0),  IK0, 256*sizeof(uint32_t));

    // ***Inputdata file to encrypt/decrypt***
    //Checking for the size of the file
    int filesize;
    filesize = fsize(filename);

    //CMS padding to have 16 bytes blocks of data
    uint8_t padElmt;
    int mod16 = filesize%16;

    padElmt = 16 - mod16; // We always add bytes for later padding detection

    //Creating required arrays
    uint8_t *inputData;
    uint8_t *outputData;
    inputData = (uint8_t*)malloc((filesize+padElmt)*sizeof(uint8_t));
    outputData = (uint8_t*)malloc((filesize+padElmt)*sizeof(uint8_t));

    //Opening the file
    FILE * inputFile;
    int result;
    inputFile = fopen(filename,"rb");

    if (inputFile == NULL) {
        perror ("Error opening file");
        exit(1);
    }
    printf("TEST\n"); //PRINTTTTTTTTTTTTT
    result = fread (inputData, sizeof(uint8_t), filesize, inputFile);
    if(result != filesize) {
        perror("Reading error from the input file");
        exit(1);
    }
    fclose(inputFile);

    //Padding
    for (int i = 0; i < padElmt; i++) {
		inputData[filesize + i] = padElmt;
    }
	filesize += padElmt; 
    printf("Filesize %d \n",filesize); //PRINTTTTTTTTTTTTT

    //Determining grid size if not given
    if(!blockNum) {
        blockNum = 1+filesize/(threadNum*16);
    }
    else {
        if(blockNum*threadNum* 16 < filesize) {
            std::cerr << std::endl <<  std::endl << "BlockNum and ThreadNum don't fit the data file ton encrypt/decrypt. ";
            exit(1);
        }
    }
    printf("Blocksize =  %d \n",blockNum);

    //Device vectors declarations and allocations
    uint32_t * devInput, * devOutput, * dev_sm_te1, * dev_sm_te2, * dev_sm_te3, * dev_sm_te4;
    uint8_t  * dev_sm_sbox;
    hipMalloc( (void **) &devInput         , filesize*sizeof(uint8_t));
    hipMalloc( (void **) &devOutput        , filesize*sizeof(uint8_t));
    hipMalloc( (void **) &dev_sm_te1       , 256*sizeof(uint32_t));
    hipMalloc( (void **) &dev_sm_te2       , 256*sizeof(uint32_t));
    hipMalloc( (void **) &dev_sm_te3       , 256*sizeof(uint32_t));
    hipMalloc( (void **) &dev_sm_te4       , 256*sizeof(uint32_t));
    hipMalloc( (void **) &dev_sm_sbox      , 256*sizeof(uint8_t));

    //Copy vectors from host memory to device memory
    if(mode) {
        hipMemcpy(dev_sm_te1       , TBox0         , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te2       , TBox1         , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te3       , TBox2         , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te4       , TBox3         , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_sbox      , sbox          , 256*sizeof(uint8_t ), hipMemcpyHostToDevice);
    }
    else {
        hipMemcpy(dev_sm_te1       , TBoxi0            , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te2       , TBoxi1            , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te3       , TBoxi2            , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te4       , TBoxi3            , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_sbox      , sbox_inv          , 256*sizeof(uint8_t ), hipMemcpyHostToDevice);
    }
    hipMemcpy(devInput, inputData, filesize*sizeof(uint8_t), hipMemcpyHostToDevice);
    
    //To record time for device execution
    hipEvent_t startDevice, stopDevice;
	hipEventCreate(&startDevice);
	hipEventCreate(&stopDevice);

    if(mode) {
        for(int i=0; i < warm_up_device ; i++) {
            encrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize, dev_sm_te1,
        dev_sm_te2, dev_sm_te3, dev_sm_te4, dev_sm_sbox);
	    }
        hipEventRecord(startDevice);
        encrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize, dev_sm_te1,
        dev_sm_te2, dev_sm_te3, dev_sm_te4, dev_sm_sbox);				
        hipEventRecord(stopDevice);
        hipEventSynchronize(stopDevice);
        hipDeviceSynchronize();
    }
    else {
        for(int i=0; i < warm_up_device ; i++) {
            decrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize, dev_sm_te1,
        dev_sm_te2, dev_sm_te3, dev_sm_te4, dev_sm_sbox);
	    }
        hipEventRecord(startDevice);						
        decrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize, dev_sm_te1,
        dev_sm_te2, dev_sm_te3, dev_sm_te4, dev_sm_sbox);
        hipEventRecord(stopDevice);
        hipEventSynchronize(stopDevice);
        hipDeviceSynchronize();
    }

    //Time calculation
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startDevice, stopDevice);
    printf("\n	GPU processing time: %f (ms)\n", milliseconds);

    //Copy results from host memory to device memory
	hipMemcpy(outputData, devOutput, filesize*sizeof(uint8_t), hipMemcpyDeviceToHost);

    //Writing results inside a file
    FILE * outputFile;
    outputFile = fopen("result.dat","wb");

    if (outputFile == NULL) {
        perror ("Error opening file");
        exit(1);
    }
    if(mode){
        result = fwrite (outputData, sizeof(uint8_t), filesize, outputFile);
        if(result != filesize) {
            perror("Writting error to the output file");
            exit(1);
        }
    }
    else {
        padElmt = outputData[filesize-padElmt-1] + padElmt;
        result = fwrite (outputData, sizeof(uint8_t), filesize-padElmt, outputFile);
        if(result != filesize-padElmt) {
            perror("Writting error to the output file");
            exit(1);
        }
    }
    fclose(outputFile);

    //Free host memory
    free(inputData);
    free(outputData);
    // Free device memory
    hipFree(devInput);
    hipFree(devOutput);
    hipFree(dev_sm_te1);
    hipFree(dev_sm_te2);
    hipFree(dev_sm_te3);
    hipFree(dev_sm_te4);
    hipFree(dev_sm_sbox);
    
	return 0;

}
