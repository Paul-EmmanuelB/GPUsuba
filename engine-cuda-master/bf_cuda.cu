#include "hip/hip_runtime.h"
// vim:foldenable:foldmethod=marker:foldmarker=[[,]]
/**
 * @version 0.1.3 (2011)
 * @author Johannes Gilger <heipei@hackvalue.de>
 * 
 * Copyright 2011 Johannes Gilger
 *
 * This file is part of engine-cuda
 *
 * engine-cuda is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License or
 * any later version.
 * 
 * engine-cuda is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with engine-cuda. If not, see <http://www.gnu.org/licenses/>.
 *
 */
#ifndef __DEVICE_EMULATION__

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <assert.h>
#include <openssl/blowfish.h>
#include <openssl/evp.h>
#include <hip/hip_runtime_api.h>
#include "cuda_common.h"
#include "common.h"

__device__ uint32_t bf_global_schedule[1042];
__shared__ uint32_t bf_schedule[1042];
__constant__ uint64_t d_iv;

__device__ uint64_t *bf_device_data;
uint8_t  *bf_host_data;

float bf_elapsed;
hipEvent_t bf_start,bf_stop;

#define BF_M  0x3fc
#define BF_0  22
#define BF_1  14
#define BF_2   6
#define BF_3  2
#define BF_ENC(LL,R,S,P) ( \
	LL^=P, \
	LL^= (((*(BF_LONG *)((unsigned char *)&(S[  0])+((R>>BF_0)&BF_M))+ \
		*(BF_LONG *)((unsigned char *)&(S[256])+((R>>BF_1)&BF_M)))^ \
		*(BF_LONG *)((unsigned char *)&(S[512])+((R>>BF_2)&BF_M)))+ \
		*(BF_LONG *)((unsigned char *)&(S[768])+((R<<BF_3)&BF_M))) \
	)

__global__ void BFencKernel(uint64_t *data) {
	register uint32_t l, r;
	register uint64_t block = data[TX];

	bf_schedule[threadIdx.x] = bf_global_schedule[threadIdx.x];
	bf_schedule[threadIdx.x+256] = bf_global_schedule[threadIdx.x+256];
	bf_schedule[threadIdx.x+512] = bf_global_schedule[threadIdx.x+512];
	bf_schedule[threadIdx.x+768] = bf_global_schedule[threadIdx.x+768];

	#if MAX_THREAD == 128
		bf_schedule[threadIdx.x+128] = bf_global_schedule[threadIdx.x+128];
		bf_schedule[threadIdx.x+384] = bf_global_schedule[threadIdx.x+384];
		bf_schedule[threadIdx.x+640] = bf_global_schedule[threadIdx.x+640];
		bf_schedule[threadIdx.x+896] = bf_global_schedule[threadIdx.x+896];
	#endif

	if(threadIdx.x < 18)
		bf_schedule[threadIdx.x+1024] = bf_global_schedule[threadIdx.x+1024];

	__syncthreads();

	register uint32_t *p,*s;

	p=bf_schedule;
	s=bf_schedule+18;

	nl2i(block, l, r);

	l^=p[0];
	BF_ENC(r,l,s,p[ 1]);
	BF_ENC(l,r,s,p[ 2]);
	BF_ENC(r,l,s,p[ 3]);
	BF_ENC(l,r,s,p[ 4]);
	BF_ENC(r,l,s,p[ 5]);
	BF_ENC(l,r,s,p[ 6]);
	BF_ENC(r,l,s,p[ 7]);
	BF_ENC(l,r,s,p[ 8]);
	BF_ENC(r,l,s,p[ 9]);
	BF_ENC(l,r,s,p[10]);
	BF_ENC(r,l,s,p[11]);
	BF_ENC(l,r,s,p[12]);
	BF_ENC(r,l,s,p[13]);
	BF_ENC(l,r,s,p[14]);
	BF_ENC(r,l,s,p[15]);
	BF_ENC(l,r,s,p[16]);
	r^=p[BF_ROUNDS+1];

	block = ((uint64_t)r) << 32 | l;
	flip64(block);
	data[TX] = block;

}

__global__ void BFdecKernel(uint64_t *data) {
	register uint32_t l, r;
	register uint64_t block = data[TX];

	bf_schedule[threadIdx.x] = bf_global_schedule[threadIdx.x];
	bf_schedule[threadIdx.x+256] = bf_global_schedule[threadIdx.x+256];
	bf_schedule[threadIdx.x+512] = bf_global_schedule[threadIdx.x+512];
	bf_schedule[threadIdx.x+768] = bf_global_schedule[threadIdx.x+768];

	#if MAX_THREAD == 128
		bf_schedule[threadIdx.x+128] = bf_global_schedule[threadIdx.x+128];
		bf_schedule[threadIdx.x+384] = bf_global_schedule[threadIdx.x+384];
		bf_schedule[threadIdx.x+640] = bf_global_schedule[threadIdx.x+640];
		bf_schedule[threadIdx.x+896] = bf_global_schedule[threadIdx.x+896];
	#endif

	if(threadIdx.x < 18)
		bf_schedule[threadIdx.x+1024] = bf_global_schedule[threadIdx.x+1024];

	__syncthreads();

	register uint32_t *p,*s;

	p=bf_schedule;
	s=bf_schedule+18;

	nl2i(block, l, r);

	l^=p[BF_ROUNDS+1];
	BF_ENC(r,l,s,p[16]);
	BF_ENC(l,r,s,p[15]);
	BF_ENC(r,l,s,p[14]);
	BF_ENC(l,r,s,p[13]);
	BF_ENC(r,l,s,p[12]);
	BF_ENC(l,r,s,p[11]);
	BF_ENC(r,l,s,p[10]);
	BF_ENC(l,r,s,p[ 9]);
	BF_ENC(r,l,s,p[ 8]);
	BF_ENC(l,r,s,p[ 7]);
	BF_ENC(r,l,s,p[ 6]);
	BF_ENC(l,r,s,p[ 5]);
	BF_ENC(r,l,s,p[ 4]);
	BF_ENC(l,r,s,p[ 3]);
	BF_ENC(r,l,s,p[ 2]);
	BF_ENC(l,r,s,p[ 1]);
	r^=p[0];

	block = ((uint64_t)r) << 32 | l;
	flip64(block);
	data[TX] = block;

	
}

__global__ void BFdecKernel_cbc(uint64_t *data, uint64_t *out) {
	register uint32_t l, r;
	register uint64_t block = data[TX];

	bf_schedule[threadIdx.x] = bf_global_schedule[threadIdx.x];
	bf_schedule[threadIdx.x+256] = bf_global_schedule[threadIdx.x+256];
	bf_schedule[threadIdx.x+512] = bf_global_schedule[threadIdx.x+512];
	bf_schedule[threadIdx.x+768] = bf_global_schedule[threadIdx.x+768];

	#if MAX_THREAD == 128
		bf_schedule[threadIdx.x+128] = bf_global_schedule[threadIdx.x+128];
		bf_schedule[threadIdx.x+384] = bf_global_schedule[threadIdx.x+384];
		bf_schedule[threadIdx.x+640] = bf_global_schedule[threadIdx.x+640];
		bf_schedule[threadIdx.x+896] = bf_global_schedule[threadIdx.x+896];
	#endif

	if(threadIdx.x < 18)
		bf_schedule[threadIdx.x+1024] = bf_global_schedule[threadIdx.x+1024];

	__syncthreads();

	register uint32_t *p,*s;

	p=bf_schedule;
	s=bf_schedule+18;

	nl2i(block, l, r);

	l^=p[BF_ROUNDS+1];
	BF_ENC(r,l,s,p[16]);
	BF_ENC(l,r,s,p[15]);
	BF_ENC(r,l,s,p[14]);
	BF_ENC(l,r,s,p[13]);
	BF_ENC(r,l,s,p[12]);
	BF_ENC(l,r,s,p[11]);
	BF_ENC(r,l,s,p[10]);
	BF_ENC(l,r,s,p[ 9]);
	BF_ENC(r,l,s,p[ 8]);
	BF_ENC(l,r,s,p[ 7]);
	BF_ENC(r,l,s,p[ 6]);
	BF_ENC(l,r,s,p[ 5]);
	BF_ENC(r,l,s,p[ 4]);
	BF_ENC(l,r,s,p[ 3]);
	BF_ENC(r,l,s,p[ 2]);
	BF_ENC(l,r,s,p[ 1]);
	r^=p[0];

	block = ((uint64_t)r) << 32 | l;
	flip64(block);

	if(blockIdx.x == 0 && threadIdx.x == 0) {
		block ^= d_iv;
	} else {
		block ^= data[TX-1];
	}

	out[TX] = block;

	
}

extern "C" void BF_cuda_transfer_key_schedule(BF_KEY *ks) {
	assert(ks);
	hipError_t cudaerrno;
	size_t ks_size = sizeof(BF_KEY);
	_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(bf_global_schedule),ks,ks_size,0,hipMemcpyHostToDevice));
}

extern "C" void BF_cuda_transfer_iv(const unsigned char *iv) {
	hipError_t cudaerrno;
	_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_iv),iv,sizeof(uint64_t),0,hipMemcpyHostToDevice));
}


extern "C" void BF_cuda_crypt(cuda_crypt_parameters *c) {

	int gridSize = c->nbytes/(MAX_THREAD*BF_BLOCK_SIZE);
	if (!(c->nbytes%(MAX_THREAD*BF_BLOCK_SIZE))==0)
		gridSize = c->nbytes/(MAX_THREAD*BF_BLOCK_SIZE)+1;

	transferHostToDevice(c->in, (uint32_t *)c->d_in, c->host_data, c->nbytes);

	if (output_verbosity==OUTPUT_VERBOSE)
		fprintf(stdout,"Starting BF kernel for %zu bytes with (%d, (%d))...\n", c->nbytes, gridSize, MAX_THREAD);

	CUDA_START_TIME

	if(c->ctx->encrypt == BF_ENCRYPT && EVP_CIPHER_CTX_mode(c->ctx) == EVP_CIPH_ECB_MODE) {
		BFencKernel<<<gridSize,MAX_THREAD>>>(c->d_in);
	} else if (!c->ctx->encrypt && EVP_CIPHER_CTX_mode(c->ctx) == EVP_CIPH_ECB_MODE) {
		BFdecKernel<<<gridSize,MAX_THREAD>>>(c->d_in);
	} else if (!c->ctx->encrypt && EVP_CIPHER_CTX_mode(c->ctx) == EVP_CIPH_CBC_MODE) {
		BFdecKernel_cbc<<<gridSize,MAX_THREAD>>>(c->d_in,c->d_out);
	}

	CUDA_STOP_TIME("BF         ")

	if(EVP_CIPHER_CTX_mode(c->ctx) == EVP_CIPH_ECB_MODE) {
		transferDeviceToHost(c->out, (uint32_t *)c->d_in, c->host_data, c->host_data, c->nbytes);
	} else {
		transferDeviceToHost(c->out, (uint32_t *)c->d_out, c->host_data, c->host_data, c->nbytes);
		BF_cuda_transfer_iv(c->in+c->nbytes-BF_BLOCK_SIZE);
	}
}
#else
#error "ERROR: DEVICE EMULATION is NOT supported."
#endif
