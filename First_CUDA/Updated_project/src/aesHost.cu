
/***************************************************************************
 *   Copyright (C) 2020                                                    *
 *                                                                         *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ***************************************************************************/


/**
	@original author Svetlin Manavski <svetlin@manavski.com>
	@updated_version author Broux Paul-Emmanuel <paulemmanuelb@gmail.com>
*/

// includes, system
#include <iostream>
#include <fstream>
//#include <stdlib.h>
//#include <stdio.h>
//#include <string.h>
//#include <math.h>


//#include <cutil.h> Outdated, we have to find an alternative to all concerned functions
//#include <hip/hip_runtime_api.h> // It can help to handle some errors
#include <helper_timer.h> // To replace timers

#include <hip/hip_runtime_api.h> //Recommended API for a lot of functions


// includes, project
#include "sbox_E.h"
#include "sbox_D.h"
#include "util.h"
#include <aesEncrypt128_kernel.h>
#include <aesDecrypt128_kernel.h>
#include <aesEncrypt256_kernel.h>
#include <aesDecrypt256_kernel.h>

using namespace std;


extern "C" void aesEncryptHandler128(unsigned *d_Result, unsigned *d_Input, int inputSize, hipEvent_t *start, hipEvent_t *stop, hipEvent_t *warm, int warm_up_device) {

	dim3  threads(BSIZE, 1);
    dim3  grid((inputSize/BSIZE)/4, 1);

	CUDA_SAFE_CALL_NO_SYNC(hipEventRecord(*warm));
	for(int i = 0; i < warm_up_device; i++){
		aesEncrypt128<<< grid, threads >>>( d_Result, d_Input, inputSize); //warmup
	}

	CUDA_SAFE_CALL_NO_SYNC(hipEventRecord(*start));
	aesEncrypt128<<< grid, threads >>>( d_Result, d_Input, inputSize); 
	CUDA_SAFE_CALL_NO_SYNC(hipEventRecord(*stop));

	CUDA_SAFE_CALL_NO_SYNC(hipEventSynchronize(*stop));
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

extern "C" void aesDecryptHandler128(unsigned *d_Result, unsigned *d_Input, int inputSize, hipEvent_t *start, hipEvent_t *stop, hipEvent_t *warm, int warm_up_device) {

	dim3  threads(BSIZE, 1);
    dim3  grid((inputSize/BSIZE)/4, 1);

	CUDA_SAFE_CALL_NO_SYNC(hipEventRecord(*warm));
	for(int i = 0; i < warm_up_device; i++){
		aesDecrypt128<<< grid, threads >>>( d_Result, d_Input, inputSize); //warmup
	}

	CUDA_SAFE_CALL_NO_SYNC(hipEventRecord(*start));
	aesDecrypt128<<< grid, threads >>>( d_Result, d_Input, inputSize); 
	CUDA_SAFE_CALL_NO_SYNC(hipEventRecord(*stop));

	CUDA_SAFE_CALL_NO_SYNC(hipEventSynchronize(*stop));
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

extern "C" void aesEncryptHandler256(unsigned *d_Result, unsigned *d_Input, int inputSize, hipEvent_t *start, hipEvent_t *stop, hipEvent_t *warm, int warm_up_device) {


	dim3  threads(BSIZE, 1);
    dim3  grid((inputSize/BSIZE)/4, 1);

	CUDA_SAFE_CALL_NO_SYNC(hipEventRecord(*warm));
	for(int i = 0; i < warm_up_device; i++){
		aesEncrypt256<<< grid, threads >>>( d_Result, d_Input, inputSize); //warmup
	}


	CUDA_SAFE_CALL_NO_SYNC(hipEventRecord(*start));
	aesEncrypt256<<< grid, threads >>>( d_Result, d_Input, inputSize); 
	CUDA_SAFE_CALL_NO_SYNC(hipEventRecord(*stop));	
	
	CUDA_SAFE_CALL_NO_SYNC(hipEventSynchronize(*stop));
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

extern "C" void aesDecryptHandler256(unsigned *d_Result, unsigned *d_Input, int inputSize, hipEvent_t *start, hipEvent_t *stop, hipEvent_t *warm, int warm_up_device) {

	
	dim3  threads(BSIZE, 1);
    dim3  grid((inputSize/BSIZE)/4, 1);

	CUDA_SAFE_CALL_NO_SYNC(hipEventRecord(*warm));
	for(int i = 0; i < warm_up_device; i++){
		aesDecrypt256<<< grid, threads >>>( d_Result, d_Input, inputSize); //warmup
	}

	CUDA_SAFE_CALL_NO_SYNC(hipEventRecord(*start));
	aesDecrypt256<<< grid, threads >>>( d_Result, d_Input, inputSize); 
	CUDA_SAFE_CALL_NO_SYNC(hipEventRecord(*stop));	
	
	CUDA_SAFE_CALL_NO_SYNC(hipEventSynchronize(*stop));
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
}


extern "C" int aesHost(unsigned char* result, const unsigned char* inData, int inputSize, const unsigned char* key, int keySize, bool toEncrypt, int bench_mode, int warm_up_device)
{
	if (inputSize < 256) 
		return -1;
	if (inputSize % 256 > 0) 
		return -11;
	if (keySize != 240 && keySize != 176) 
		return -2;
	if (!result || !inData || !key)
		return -3;

    int deviceCount;                                                         
    CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));            
    if (deviceCount == 0) {                                                  
        fprintf(stderr, "There is no device.\n");                            
        exit(EXIT_FAILURE);                                                  
    }                                                                        
    int dev;                                                                 
    for (dev = 0; dev < deviceCount; ++dev) {                                
        hipDeviceProp_t deviceProp;                                           
        CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&deviceProp, dev));

        if (deviceProp.major >= 1)                                           
            break;                                                           
    }                                                                        
    if (dev == deviceCount) {                                                
        fprintf(stderr, "There is no device supporting CUDA.\n");            
        exit(EXIT_FAILURE);                                                  
    }                                                                        
    else                                                                     
        CUDA_SAFE_CALL(hipSetDevice(dev));                                  


    // allocate device memory
    unsigned * d_Input;
    CUDA_SAFE_CALL( hipMalloc((void**) &d_Input, inputSize) );

	// the size of the memory for the key must be equal to keySize (every thread copies one key byte to shared memory)
    unsigned * d_Key;
    CUDA_SAFE_CALL( hipMalloc((void**) &d_Key, keySize) );

	/*
	//Outdated code
	unsigned int ext_timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&ext_timer));
    CUT_SAFE_CALL(cutStartTimer(ext_timer));
	*/
	//We replace it with following lines 
	StopWatchInterface *ext_timer = NULL;
	CUT_SAFE_CALL( sdkCreateTimer(&ext_timer) );
	CUT_SAFE_CALL( sdkStartTimer(&ext_timer) );

	hipEvent_t startHost, stopHost;
	CUDA_SAFE_CALL_NO_SYNC(hipEventCreate(&startHost));
	CUDA_SAFE_CALL_NO_SYNC(hipEventCreate(&stopHost));
	CUDA_SAFE_CALL_NO_SYNC(hipEventRecord(startHost));


    // copy host memory to device
    CUDA_SAFE_CALL( hipMemcpy(d_Input, inData, inputSize, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy(d_Key, key, keySize, hipMemcpyHostToDevice) );

	//texture
	hipChannelFormatDesc chDesc;
	chDesc.x = 32;
	chDesc.y = 0;
	chDesc.z = 0;
	chDesc.w = 0;
	chDesc.f = hipChannelFormatKindUnsigned;
	texEKey.normalized = false;
	texDKey.normalized = false;
	texEKey128.normalized = false;
	texDKey128.normalized = false;

	CUDA_SAFE_CALL( hipBindTexture( 0, &texEKey128, d_Key, &chDesc, (size_t)keySize) );
	CUDA_SAFE_CALL( hipBindTexture( 0, &texDKey128, d_Key, &chDesc, (size_t)keySize) );
	CUDA_SAFE_CALL( hipBindTexture( 0, &texEKey, d_Key, &chDesc, (size_t)keySize) );
	CUDA_SAFE_CALL( hipBindTexture( 0, &texDKey, d_Key, &chDesc, (size_t)keySize) );

    // allocate device memory for result
    unsigned int size_Result = inputSize;
    unsigned * d_Result;
    CUDA_SAFE_CALL( hipMalloc((void**) &d_Result, size_Result) );
	CUDA_SAFE_CALL( hipMemset(d_Result, 0, size_Result) );
	
	/*
	// Outdated
	unsigned int int_timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&int_timer));
    CUT_SAFE_CALL(cutStartTimer(int_timer));
	*/
	//We replace it with following lines for timing.	
	hipEvent_t startDevice, stopDevice, warmUp;
	CUDA_SAFE_CALL_NO_SYNC(hipEventCreate(&startDevice));
	CUDA_SAFE_CALL_NO_SYNC(hipEventCreate(&stopDevice));
	CUDA_SAFE_CALL_NO_SYNC(hipEventCreate(&warmUp));


	//Encryption or decryption
	if (!toEncrypt) {
		if(!bench_mode)
			printf("\nDECRYPTION.....\n\n");
		
		if (keySize != 240)
			aesDecryptHandler128( d_Result, d_Input, inputSize, &startDevice, &stopDevice, &warmUp, warm_up_device);
		else
			aesDecryptHandler256( d_Result, d_Input, inputSize, &startDevice, &stopDevice, &warmUp, warm_up_device);
	
	} else {
		if(!bench_mode)
			printf("\nENCRYPTION.....\n\n");

		if (keySize != 240)
			aesEncryptHandler128( d_Result, d_Input, inputSize, &startDevice, &stopDevice, &warmUp, warm_up_device);
		else
			aesEncryptHandler256( d_Result, d_Input, inputSize, &startDevice, &stopDevice, &warmUp, warm_up_device);
	}
	
	/*
	//Outdated
	CUT_SAFE_CALL(cutStopTimer(int_timer));
    printf("GPU processing time: %f (ms)\n", cutGetTimerValue(int_timer));
    CUT_SAFE_CALL(cutDeleteTimer(int_timer));
	*/
	//Replacement with followings lines fot timing.
	float milliseconds = 0;
	float millisecondsWarmUp = 0;	

	std::ofstream benchFile;
	if(bench_mode) {
		
		benchFile.open("bench/benchResult", ios::app);

		hipEventElapsedTime(&milliseconds, startDevice, stopDevice);
		hipEventElapsedTime(&millisecondsWarmUp, warmUp, startDevice);
		printf("	GPU processing time: %f (ms)\n", milliseconds);
		benchFile << "	BENCH RESULT for device's warm up of " << warm_up_device << std::endl;
		benchFile << "	GPU kernel execution  (ms)	" << milliseconds << std::endl;
	}

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

    // copy result from device to host
    CUDA_SAFE_CALL(hipMemcpy(result, d_Result, size_Result, hipMemcpyDeviceToHost) );
	

	/*
	//Outdated
    CUT_SAFE_CALL(cutStopTimer(ext_timer));
    printf("Total processing time: %f (ms)\n\n", cutGetTimerValue(ext_timer));
    CUT_SAFE_CALL(cutDeleteTimer(ext_timer));
	*/
	//Replacement with followings lines for timing.
	if(bench_mode) {
		CUDA_SAFE_CALL_NO_SYNC(hipEventRecord(stopHost));
		CUDA_SAFE_CALL_NO_SYNC(hipEventSynchronize(stopHost));
		hipEventElapsedTime(&milliseconds, startHost, stopHost);
		
	
		float totalProcessingTime = milliseconds - millisecondsWarmUp;
		printf("	Total processing time : %f (ms)\n\n", totalProcessingTime);
		//printf("	Warmup: %f (ms)\n\n", millisecondsWarmUp);
		benchFile << "	Total processing time  (ms)	" << totalProcessingTime << std::endl << std::endl;
	}

	//Deleting Clocks
	CUDA_SAFE_CALL_NO_SYNC(hipEventDestroy(startDevice));
	CUDA_SAFE_CALL_NO_SYNC(hipEventDestroy(stopDevice));
	CUDA_SAFE_CALL_NO_SYNC(hipEventDestroy(warmUp));
	CUDA_SAFE_CALL_NO_SYNC(hipEventDestroy(startHost));
	CUDA_SAFE_CALL_NO_SYNC(hipEventDestroy(stopHost));

    // cleanup memory
    CUDA_SAFE_CALL(hipFree(d_Input));
    CUDA_SAFE_CALL(hipFree(d_Key));
    CUDA_SAFE_CALL(hipFree(d_Result));

    return 0;
}

