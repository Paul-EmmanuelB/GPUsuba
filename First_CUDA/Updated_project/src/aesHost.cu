
/***************************************************************************
 *   Copyright (C) 2006                                                    *
 *                                                                         *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ***************************************************************************/


/**
	@original author Svetlin Manavski <svetlin@manavski.com>
	@updated_version author Broux Paul-Emmanuel <paulemmanuelb@gmail.com>
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>


//#include <cutil.h> Outdated, we have to find an alternative to all concerned functions
//#include <hip/hip_runtime_api.h> // It can help to handle some errors
#include <helper_timer.h> // To replace timers

#include <hip/hip_runtime_api.h> //Recommended API for a lot of functions


// includes, project
#include "sbox_E.h"
#include "sbox_D.h"
#include "util.h"
#include <aesEncrypt128_kernel.h>
#include <aesDecrypt128_kernel.h>
#include <aesEncrypt256_kernel.h>
#include <aesDecrypt256_kernel.h>


#define BENCH_ON




extern "C" void aesEncryptHandler128(unsigned *d_Result, unsigned *d_Input, int inputSize) {

	dim3  threads(BSIZE, 1);
    dim3  grid((inputSize/BSIZE)/4, 1);

	aesEncrypt128<<< grid, threads >>>( d_Result, d_Input, inputSize);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

extern "C" void aesDecryptHandler128(unsigned *d_Result, unsigned *d_Input, int inputSize) {

	dim3  threads(BSIZE, 1);
    dim3  grid((inputSize/BSIZE)/4, 1);

	aesDecrypt128<<< grid, threads >>>( d_Result, d_Input, inputSize);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

extern "C" void aesEncryptHandler256(unsigned *d_Result, unsigned *d_Input, int inputSize) {

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3  threads(BSIZE, 1);
    dim3  grid((inputSize/BSIZE)/4, 1);


	for(int i = 0; i < 10; i++){
		aesEncrypt256<<< grid, threads >>>( d_Result, d_Input, inputSize); //warmup
	}


	hipEventRecord(start);
	aesEncrypt256<<< grid, threads >>>( d_Result, d_Input, inputSize); 
	hipEventRecord(stop);	
	
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
	
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("GPU, other method processing time: %f (ms)\n", milliseconds);

	hipEventDestroy(start);
	hipEventDestroy(stop);

}

extern "C" void aesDecryptHandler256(unsigned *d_Result, unsigned *d_Input, int inputSize) {

	dim3  threads(BSIZE, 1);
    dim3  grid((inputSize/BSIZE)/4, 1);

	aesDecrypt256<<< grid, threads >>>( d_Result, d_Input, inputSize);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
}


extern "C" int aesHost(unsigned char* result, const unsigned char* inData, int inputSize, const unsigned char* key, int keySize, bool toEncrypt)
{
	if (inputSize < 256) 
		return -1;
	if (inputSize % 256 > 0) 
		return -11;
	if (keySize != 240 && keySize != 176) 
		return -2;
	if (!result || !inData || !key)
		return -3;

    int deviceCount;                                                         
    CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));            
    if (deviceCount == 0) {                                                  
        fprintf(stderr, "There is no device.\n");                            
        exit(EXIT_FAILURE);                                                  
    }                                                                        
    int dev;                                                                 
    for (dev = 0; dev < deviceCount; ++dev) {                                
        hipDeviceProp_t deviceProp;                                           
        CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&deviceProp, dev));
		
		/*
		//Device informations
		printf("Device Number: %d\n", dev);
		printf("  Device name: %s\n", deviceProp.name);
		printf("  Memory Clock Rate (KHz): %d\n",
		       deviceProp.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
		       deviceProp.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
		       2.0*deviceProp.memoryClockRate*(deviceProp.memoryBusWidth/8)/1.0e6);
		printf("  Total global memory (bytes): %ld\n",
		       deviceProp.totalGlobalMem);
		printf("  Total constant memory (bytes): %d\n",
		       deviceProp.totalConstMem);
		*/

        if (deviceProp.major >= 1)                                           
            break;                                                           
    }                                                                        
    if (dev == deviceCount) {                                                
        fprintf(stderr, "There is no device supporting CUDA.\n");            
        exit(EXIT_FAILURE);                                                  
    }                                                                        
    else                                                                     
        CUDA_SAFE_CALL(hipSetDevice(dev));                                  


    // allocate device memory
    unsigned * d_Input;
    CUDA_SAFE_CALL( hipMalloc((void**) &d_Input, inputSize) );

	// the size of the memory for the key must be equal to keySize (every thread copies one key byte to shared memory)
    unsigned * d_Key;
    CUDA_SAFE_CALL( hipMalloc((void**) &d_Key, keySize) );

	/*
	//Outdated code
	unsigned int ext_timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&ext_timer));
    CUT_SAFE_CALL(cutStartTimer(ext_timer));
	*/
	//We replace it with following lines 
	StopWatchInterface *ext_timer = NULL;
	CUT_SAFE_CALL( sdkCreateTimer(&ext_timer) );
	CUT_SAFE_CALL( sdkStartTimer(&ext_timer) );


    // copy host memory to device
    CUDA_SAFE_CALL( hipMemcpy(d_Input, inData, inputSize, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy(d_Key, key, keySize, hipMemcpyHostToDevice) );

	//texture
	hipChannelFormatDesc chDesc;
	chDesc.x = 32;
	chDesc.y = 0;
	chDesc.z = 0;
	chDesc.w = 0;
	chDesc.f = hipChannelFormatKindUnsigned;
	texEKey.normalized = false;
	texDKey.normalized = false;
	texEKey128.normalized = false;
	texDKey128.normalized = false;

	CUDA_SAFE_CALL( hipBindTexture( 0, &texEKey128, d_Key, &chDesc, (size_t)keySize) );
	CUDA_SAFE_CALL( hipBindTexture( 0, &texDKey128, d_Key, &chDesc, (size_t)keySize) );
	CUDA_SAFE_CALL( hipBindTexture( 0, &texEKey, d_Key, &chDesc, (size_t)keySize) );
	CUDA_SAFE_CALL( hipBindTexture( 0, &texDKey, d_Key, &chDesc, (size_t)keySize) );

    // allocate device memory for result
    unsigned int size_Result = inputSize;
    unsigned * d_Result;
    CUDA_SAFE_CALL( hipMalloc((void**) &d_Result, size_Result) );
	CUDA_SAFE_CALL( hipMemset(d_Result, 0, size_Result) );
	
	/*
	// Outdated
	unsigned int int_timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&int_timer));
    CUT_SAFE_CALL(cutStartTimer(int_timer));
	*/
	//We replace it with following lines 
	StopWatchInterface *int_timer = NULL;
	CUT_SAFE_CALL( sdkCreateTimer(&int_timer) );
	CUT_SAFE_CALL( sdkStartTimer(&int_timer) );

	/*
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);
	*/

	if (!toEncrypt) {
		#ifndef BENCH_ON
			printf("\nDECRYPTION.....\n\n");
		#endif //#ifndef BENCH_ON
		if (keySize != 240)
			aesDecryptHandler128( d_Result, d_Input, inputSize);
		else
			aesDecryptHandler256( d_Result, d_Input, inputSize);
	} else {
		#ifndef BENCH_ON
			printf("\nENCRYPTION.....\n\n");
		#endif //#ifndef BENCH_ON
		if (keySize != 240)
			aesEncryptHandler128( d_Result, d_Input, inputSize);
		else
			aesEncryptHandler256( d_Result, d_Input, inputSize);
	}
	
	/*
	//Outdated
	CUT_SAFE_CALL(cutStopTimer(int_timer));
    printf("GPU processing time: %f (ms)\n", cutGetTimerValue(int_timer));
    CUT_SAFE_CALL(cutDeleteTimer(int_timer));
	*/
	//Replacement with followings lines
	CUT_SAFE_CALL(sdkStopTimer(&int_timer));
    printf("GPU processing time: %f (ms)\n", sdkGetTimerValue(&int_timer));
    CUT_SAFE_CALL(sdkDeleteTimer(&int_timer));
	
	//hipEventRecord(stop);

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

    // copy result from device to host
    CUDA_SAFE_CALL(hipMemcpy(result, d_Result, size_Result, hipMemcpyDeviceToHost) );
	

	/*hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("GPU, other method processing time: %f (ms)\n", milliseconds);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	*/

	/*
	//Outdated
    CUT_SAFE_CALL(cutStopTimer(ext_timer));
    printf("Total processing time: %f (ms)\n\n", cutGetTimerValue(ext_timer));
    CUT_SAFE_CALL(cutDeleteTimer(ext_timer));
	*/
	//Replacement with followings lines
	CUT_SAFE_CALL(sdkStopTimer(&ext_timer));
    printf("Total processing time: %f (ms)\n\n", sdkGetTimerValue(&ext_timer));
    CUT_SAFE_CALL(sdkDeleteTimer(&ext_timer));

    // cleanup memory
    CUDA_SAFE_CALL(hipFree(d_Input));
    CUDA_SAFE_CALL(hipFree(d_Key));
    CUDA_SAFE_CALL(hipFree(d_Result));

    return 0;
}

