#include "hip/hip_runtime.h"
/*

From Jia, Z., Maggioni, M., Smith, J., & Scarpazza, D. P. (2019). Dissecting the NVidia Turing T4 GPU via Microbenchmarking

*/



__global__ void KernelFunction() {
    //declare shared memory space
    __shared__ unsigned int s_tvalue[];
    __shared__ undigned int s_index[];
    //preheat the data
    for(it=0; it<iterations; it++) {
        start_time=clock();
        j=my_array[j];
        //store the array index
        s_index[it]=j;
        end_time=clock();
        //store the access latency
        s_tvalue[it]=end_time-start_time;
    }
}