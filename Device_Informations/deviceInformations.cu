/*

Print all basic device informations you could need. 

*/

#include <cstdio>
#include <hip/hip_runtime_api.h>



// To prevent calls from errors
#define CUDA_SAFE_CALL_NO_SYNC(x)                                			\
  do {                                                            			\
    hipError_t err = x;			                              			\
    if (err != hipSuccess) {                                 	  			\
		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",       \
		            __FILE__, __LINE__, hipGetErrorString( err) );         \
		exit(EXIT_FAILURE);                                                 \
    }                                                             			\
  } while(0)




void deviceInfo() {

	int deviceCount;
	hipDeviceProp_t deviceProp;
                                                     
    CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));          
    if (deviceCount == 0) {                                                  
        fprintf(stderr, "There is no device.\n");                            
        exit(EXIT_FAILURE);
	}
	
	for(int dev=0 ; dev < deviceCount ; dev++){
		CUDA_SAFE_CALL_NO_SYNC( hipGetDeviceProperties(&deviceProp, dev) );
		printf("	Device Number:						%d\n"					, dev);
		printf("	Name: 							%s\n"						, deviceProp.name);
		//printf("	Unique identifier 							%lu\n"		, deviceProp.uuid);
		printf("	Total Global memory (bytes)				%-lu\n"				, deviceProp.totalGlobalMem);
		printf("	Shared memory per block (bytes)				%-10lu\n"		, deviceProp.sharedMemPerBlock);		
		printf("	Maximum 32-bits registers per block			%d\n"			, deviceProp.regsPerBlock);
		printf("	Warp size (threads)					%d\n"					, deviceProp.warpSize);
		printf("	Max threads per block					%d\n"				, deviceProp.maxThreadsPerBlock);
		printf("	Max threads in x dimension				%d\n"				, deviceProp.maxThreadsDim[0]);
		printf("	Max threads in y dimension				%d\n"				, deviceProp.maxThreadsDim[1]);
		printf("	Max threads in z dimension				%d\n"				, deviceProp.maxThreadsDim[2]);
		printf("	Where x*y*z					<=	1024\n"		);
		printf("	Max grid size (block)					%d\n"				, deviceProp.maxGridSize[0]);
		printf("	Max grid size (block)					%d\n"				, deviceProp.maxGridSize[1]);
		printf("	Max grid size (block)					%d\n"				, deviceProp.maxGridSize[2]);
		printf("	Clock rate (kHZ)					%d\n"					, deviceProp.clockRate);
		printf("	Total constant memory (bytes)				%lu\n"			, deviceProp.totalConstMem);
		printf("	Major							%d\n"						, deviceProp.major);
		printf("	Minor							%d\n"						, deviceProp.minor);
		printf("	Multiprocessor count					%d\n"				, deviceProp.multiProcessorCount);
		printf("	Run time limit for kernel execution			%d\n"			, deviceProp.kernelExecTimeoutEnabled );
		printf("	Integrated GPU (motherboard)				%d\n"			, deviceProp.integrated);
		printf("	Mapping host mem into CUDA adress space			%d\n"		, deviceProp.canMapHostMemory);
		printf("	Compute mode						%d\n"					, deviceProp.computeMode);
		printf("	Memory Clock Rate (KHz): 				%d\n"				, deviceProp.memoryClockRate);
		printf("	Memory Bus Width (bits): 				%d\n"				, deviceProp.memoryBusWidth);
		printf("	Peak Memory Bandwidth (GB/s): 				%f\n"			, 2.0*deviceProp.memoryClockRate*(deviceProp.memoryBusWidth/8)/1.0e6); //Double rate memory explain the x2
		printf("	L2 cache size (bytes)					%d\n"				, deviceProp.l2CacheSize);
		printf("	Max resident threads per multiprocessor			%d\n"		, deviceProp.maxThreadsPerMultiProcessor);
		printf("	Supports caching globals in L1				%d\n"			, deviceProp.globalL1CacheSupported);
		printf("	Supports caching locals in L1				%d\n"			, deviceProp.localL1CacheSupported);
		printf("	Shared memory per MP (bytes)				%lu\n"			, deviceProp.sharedMemPerMultiprocessor); //Shared by all threads blocks simultaneously resident on a multiprocessor
		printf("	32-bits registers per MP				%d\n"				, deviceProp.regsPerMultiprocessor); //Shared by all threads blocks simultaneously resident on a multiprocessor


	}	

}



int main(int argc, char *argv[]){
	
	deviceInfo();	

	return 0;
}

