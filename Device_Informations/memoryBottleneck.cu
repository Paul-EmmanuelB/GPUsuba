#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>

#include "utils.h"

typedef unsigned int uint32;

__global__ static void memoryTransfer( uint32* dev_input, uint32* dev_output, 
size_t inputSize)
{

    // Index calculations
    int x           = blockIdx.x * blockDim.x + threadIdx.x;    //global x id
    int y           = blockIdx.y * blockDim.y + threadIdx.y;    //global y id
    int xwidth      = blockDim.x * gridDim.x;                   //X width of the grid
    int global_tid  = y*xwidth + x;

    uint32 w1, w2, w3, w4;

    if(global_tid < inputSize/16) {
        w1 = dev_input[4*global_tid];
        w2 = dev_input[4*global_tid+1];
        w3 = dev_input[4*global_tid+2];
        w4 = dev_input[4*global_tid+3];

        dev_output[4*global_tid  ] = w1;
        dev_output[4*global_tid+1] = w2;
        dev_output[4*global_tid+2] = w3;
        dev_output[4*global_tid+3] = w4;
    }

}


__global__ static void encrypt_Kernel( size_t inputSize)
{
    // Index calculations
    int tid         = threadIdx.y*blockDim.x + threadIdx.x;     //local id
    int x           = blockIdx.x * blockDim.x + threadIdx.x;    //global x id
    int y           = blockIdx.y * blockDim.y + threadIdx.y;    //global y id
    int w           = blockDim.x * gridDim.x;                   //width of the grid
    int global_tid  = y*w + x;                                  //global id

    int blockSize = blockDim.x * blockDim.y; 
    
    uint32_t w1,w2,w3,w4,s1,s2,s3,s4;
    int ROUNDS = 11;

    // store the T-boxes and sbox in shared memory.
    __shared__ uint32_t sm_te1[256], sm_te2[256], sm_te3[256], sm_te4[256];
    __shared__ uint8_t sm_sbox[256];
    
    // Loading shared memory. 256 elements are needed
    int elemPerThread = 256/blockSize;
    
    if ( !elemPerThread && tid<256) {
        //load dev_sm_te1, dev_sm_te2, dev_sm_te3, dev_sm_te4 and
        // sm_sbox to share memory variables sm_te1, sm_te2,
        //sm_te3, sm_te4 and sm_sbox;
        sm_te1[tid]   = tid;
        sm_te2[tid]   = tid;
        sm_te3[tid]   = tid;
        sm_te4[tid]   = tid;
        sm_sbox[tid]  = tid;
    }
    else {
        for(int i=0; i<elemPerThread; i++) {
            sm_te1[tid*elemPerThread  + i]   = tid*(elemPerThread+1) + i;
            sm_te2[tid*elemPerThread  + i]   = tid*(elemPerThread+1) + i;
            sm_te3[tid*elemPerThread  + i]   = tid*(elemPerThread+1) + i;
            sm_te4[tid*elemPerThread  + i]   = tid*(elemPerThread+1) + i;
            sm_sbox[tid*elemPerThread + i]   = tid*(elemPerThread+1) + i;
        }
        int modEPT = 256%blockSize; //256 is not a multiple of blockSize
        if(!modEPT && (tid == blockSize-1)) {
            for(int i=0; i<modEPT; i++) {
                sm_te1[tid*(elemPerThread+1)  + i]   = tid*(elemPerThread+1) + i;
                sm_te2[tid*(elemPerThread+1)  + i]   = tid*(elemPerThread+1) + i;
                sm_te3[tid*(elemPerThread+1)  + i]   = tid*(elemPerThread+1) + i;
                sm_te4[tid*(elemPerThread+1)  + i]   = tid*(elemPerThread+1) + i;
                sm_sbox[tid*(elemPerThread+1) + i]   = tid*(elemPerThread+1) + i;
            }
        }
    }
    __syncthreads();

    // Each thread treat 16 bytes. 
    if(global_tid < inputSize/16) {
        
        //load the cipher blocks, all the global memory transactions are
        //coalesced. The original plain text load from files, due to the read
        //procedure reverse the byte order of the 32-bit words, So a reverse
        //process was necessary.


        w1 = 4*global_tid;
        w2 = 4*global_tid+1;
        w3 = 4*global_tid+2;
        w4 = 4*global_tid+3;

        // First round AddRoundKey: ex-or with round key
        //w1 ^= const_expkey[0];
        //w2 ^= const_expkey[1];
        //w3 ^= const_expkey[2];
        //w4 ^= const_expkey[3];

        // Round transformation: a set of table lookups operations.
        #pragma unroll
        for (int i = 1; i < 10; i++) {
            s1 = (sm_te4[(w4 >> 24)] ^ sm_te3[(w3 >> 16) & 0xFF] ^ sm_te2[(w2 >> 8) & 0xFF] ^ sm_te1[w1 & 0xFF]);
            s2 = (sm_te4[(w1 >> 24)] ^ sm_te3[(w4 >> 16) & 0xFF] ^ sm_te2[(w3 >> 8) & 0xFF] ^ sm_te1[w2 & 0xFF]);
            s3 = (sm_te4[(w2 >> 24)] ^ sm_te3[(w1 >> 16) & 0xFF] ^ sm_te2[(w4 >> 8) & 0xFF] ^ sm_te1[w3 & 0xFF]);
            s4 = (sm_te4[(w3 >> 24)] ^ sm_te3[(w2 >> 16) & 0xFF] ^ sm_te2[(w1 >> 8) & 0xFF] ^ sm_te1[w4 & 0xFF]);

            w1 = s1 ^ const_expkey[i * 4];
            w2 = s2 ^ const_expkey[i * 4 + 1];
            w3 = s3 ^ const_expkey[i * 4 + 2];
            w4 = s4 ^ const_expkey[i * 4 + 3];	
        } 

        // The final round doesn’t include the MixColumns
        s1  = (uint32_t)(sm_sbox[ w1        & 0xFF]);
        s1 |= (uint32_t)(sm_sbox[(w2 >>  8) & 0xFF]) << 8;
        s1 |= (uint32_t)(sm_sbox[(w3 >> 16) & 0xFF]) << 16;
        s1 |= (uint32_t)(sm_sbox[(w4 >> 24)       ]) << 24; //SubBytes and ShiftRows
        s1 ^= const_expkey[(ROUNDS - 1) * 4]; //AddRoundKey

        
        s2  = (uint32_t)(sm_sbox[ w2        & 0xFF]);
        s2 |= (uint32_t)(sm_sbox[(w3 >>  8) & 0xFF]) << 8;
        s2 |= (uint32_t)(sm_sbox[(w4 >> 16) & 0xFF]) << 16;
        s2 |= (uint32_t)(sm_sbox[(w1 >> 24)       ]) << 24; //SubBytes and ShiftRows
        s2 ^= const_expkey[(ROUNDS - 1) * 4 + 1]; //AddRoundKey

        s3  = (uint32_t)(sm_sbox[ w3        & 0xFF]);
        s3 |= (uint32_t)(sm_sbox[(w4 >>  8) & 0xFF]) << 8;
        s3 |= (uint32_t)(sm_sbox[(w1 >> 16) & 0xFF]) << 16;
        s3 |= (uint32_t)(sm_sbox[(w2 >> 24)       ]) << 24; //SubBytes and ShiftRows
        s3 ^= const_expkey[(ROUNDS - 1) * 4 + 2]; //AddRoundKey

        s4  = (uint32_t)(sm_sbox[ w4        & 0xFF]);
        s4 |= (uint32_t)(sm_sbox[(w1 >>  8) & 0xFF]) << 8;
        s4 |= (uint32_t)(sm_sbox[(w2 >> 16) & 0xFF]) << 16;
        s4 |= (uint32_t)(sm_sbox[(w3 >> 24)       ]) << 24; //SubBytes and ShiftRows
        s4 ^= const_expkey[(ROUNDS - 1) * 4 + 3]; //AddRoundKey
    }
}

int main(int argc, char * argv[]) {

    ///////////////////////////////////////////////////////////////
    // command line arguments
    ///////////////////////////////////////////////////////////////
    char *  filename;
    int     threadNum           = 512;  // Threads per block. This is a recommanded number.
    int     blockNum            = 0;    // Number of blocks in the grid

    std::cout << std::endl << "********************************************************************" ;
    std::cout << std::endl << "******                   CUDA Tests                           ******" ;
    std::cout << std::endl << "******                Memory Bottleneck                       ******" ;
    std::cout << std::endl << "********************************************************************" << std::endl << std::endl; 

    if (argc > 1){
        for( int n=1 ; n<argc ; n=n+2 ) {
            if((strcmp(argv[n],"-filename") == 0) && (n+1<argc)) {
                filename = argv[n+1];
            }
            else if((strcmp(argv[n],"-threadNum") == 0) && (n+1<argc)) {
                threadNum = atoi(argv[n+1]);
                if(threadNum ==0) {
                    printf("\n threadNum must be a non-null value.\n");
                    exit(1);
                }
            }
            else if((strcmp(argv[n],"-blockNum") == 0) && (n+1<argc)) {
                blockNum = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-help") == 0)) {
                std::cout << "   This is a CUDA test program." << std::endl;
				std::cout << "   \"-options value\" availables are:" << std::endl;
                std::cout << "   -filename, the file path to encrypt or decrypt." << std::endl;
                std::cout << "   -threadNum to set the number of threads per block. Default recommended value is 512." << std::endl;
                std::cout << "   -blockNum to set the number of blocks in the grid. Default value is 0 and will create enough blocks taking into account the input file size and the threadNum argument." << std::endl;
                std::cout << "   The order between options is not important." << std::endl << std::endl;
                exit(0);
            }
            else {
                std::cout << std::endl << "Argument " << argv[n] << " does not correspond to any valid arguments. Type -help for details about valid command line arguments." <<std::endl;
		        exit(1);
            }
        }


    }
	else {
		std::cout << std::endl << std::endl << "Not enough arguments. Type -help option in the command line for further explanations." << std::endl;
		exit(1);
	}

    std::cout << "    threadNum		= " << threadNum        << std::endl;
    std::cout << "    blockNum		= " << blockNum         << std::endl;
    std::cout << "    Filename		= " << filename  << std::endl << std::endl;

    // ***Inputdata file to encrypt/decrypt***
    //Checking for the size of the file
    int filesize;
    filesize = fsize(filename);
    uint8_t *inputData, *outputData;
    inputData = (uint8_t*)malloc((filesize)*sizeof(uint8_t));
    outputData = (uint8_t*)malloc((filesize)*sizeof(uint8_t));

    //Opening the file
    FILE * inputFile;
    int result;
    inputFile = fopen(filename,"rb");

    if (inputFile == NULL) {
        perror ("Error opening file");
        exit(1);
    }
    result = fread (inputData, sizeof(uint8_t), filesize, inputFile);
    if(result != filesize) {
        perror("Reading error from the input file");
        exit(1);
    }
    fclose(inputFile);


    if(!blockNum) {
        blockNum = 1+filesize/(threadNum);
    }

    std::cout << "    Gridsize in term of block: " << blockNum  << std::endl;


    //Device vectors declarations and allocations
    uint32 * devInput, * devOutput;
    hipMalloc( (void **) &devInput         , filesize*sizeof(uint8_t));
    hipMalloc( (void **) &devOutput        , filesize*sizeof(uint8_t));


    //To record the device time execution
    hipEvent_t startDevice, stopDevice;
	checkCudaErrors(hipEventCreate(&startDevice));
	checkCudaErrors(hipEventCreate(&stopDevice));

    //checkCudaErrors(hipEventRecord(startHost, NULL));
    hipMemcpy(devInput, inputData, filesize*sizeof(uint8_t), hipMemcpyHostToDevice);

    //Warm Up
    for(int i=0; i < 1000 ; i++) { 
        encrypt_Kernel<<<blockNum,threadNum>>>(filesize);
    }
    
    checkCudaErrors(hipEventRecord(startDevice, NULL));
    for(int j=0; j<1000; j++){ //for benchmarking
        encrypt_Kernel<<<blockNum,threadNum>>>(filesize);
    }			
    checkCudaErrors(hipEventRecord(stopDevice, NULL));

    //Copy results from host memory to device memory
	hipMemcpy(outputData, devOutput, filesize*sizeof(uint8_t), hipMemcpyDeviceToHost);

    checkCudaErrors(hipEventSynchronize(stopDevice));


    //Time calculation
    float Devmsec   = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&Devmsec, startDevice, stopDevice));
    Devmsec /= 1000;
    double throughput = 1.0e-9f*8*filesize/(Devmsec*1.0e-3f);
    printf("\n	GPU processing time: %f (ms)", Devmsec);
    printf("\n	GPU throughput: %f (Gbps)\n", throughput);


    //Free host memory
    free(inputData);
    free(outputData);
    // Free device memory
    hipFree(devInput);
    hipFree(devOutput);

}