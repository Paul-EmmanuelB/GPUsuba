
/***************************************************************************
 *   Copyright (C) 2020                                                    *
 *                                                                         *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ***************************************************************************/


/**
	@author Broux Paul-Emmanuel <paulemmanuelb@gmail.com>
 */

#define BENCH_ON

#include <stdio.h>
#include <cstdlib>

#include "utils.h"
#include "typedef.h"
#include "encryptKernelECB.h"
#include "decryptKernelECB.h"
#include "sboxE.h"
#include "sboxD.h"


int main(int argc, char * argv[]) {


    ///////////////////////////////////////////////////////////////
    // command line arguments
    ///////////////////////////////////////////////////////////////
    int     warm_up_device      = 0;    // GPU kernel warm up
    int     threadNum           = 512;  // Threads per block. This is a recommanded number.
    int     blockNum            = 0;    // Number of blocks in the grid
    int     mode                = 1;    // Encryption mode, 1 to encrypt or 0 to decrypt.
    char *  filename;
    char *  keyfilename; 

    std::cout << std::endl << "********************************************************************" ;
    std::cout << std::endl << "******                   AES-128 CUDA                         ******" ;
    std::cout << std::endl << "******                     Reference                          ******" ;
    std::cout << std::endl << "********************************************************************" << std::endl << std::endl; 

    if (argc > 1){
        for( int n=1 ; n<argc ; n=n+2 ) {
            if((strcmp(argv[n],"-wuDevice") == 0) && (n+1<argc)) {
                warm_up_device = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-threadNum") == 0) && (n+1<argc)) {
                threadNum = atoi(argv[n+1]);
                if(threadNum ==0) {
                    printf("\n threadNum must be a non-null value.\n");
                    exit(1);
                }
            }
            else if((strcmp(argv[n],"-blockNum") == 0) && (n+1<argc)) {
                blockNum = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-mode") == 0) && (n+1<argc)) {
                mode = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-filename") == 0) && (n+1<argc)) {
                filename = argv[n+1];
            }
            else if((strcmp(argv[n],"-keyfilename") == 0) && (n+1<argc)) {
                keyfilename = argv[n+1];
            }
            else if((strcmp(argv[n],"-help") == 0)) {
                std::cout << "   This is a AES-128 implementation." << std::endl;
				std::cout << "   \"-options value\" availables are:" << std::endl;
                std::cout << "   -mode, 1 to encrypt and 0 to decrypt. Default value is 1." << std::endl;
                std::cout << "   -filename, the file path to encrypt or decrypt." << std::endl;
                std::cout << "   -keyfilename, the 128 key file's path to use for encyption or decryption." << std::endl;
                std::cout << "   -threadNum to set the number of threads per block. Default recommended value is 512." << std::endl;
                std::cout << "   -blockNum to set the number of blocks in the grid. Default value is 0 and will create enough blocks taking into account the input file size and the threadNum argument." << std::endl;
                std::cout << "   -wuDevice number of device kernel launch before timing (around 1000 or less is adviced) " << std::endl;
                std::cout << "   The order between options is not important." << std::endl << std::endl;
                exit(0);
            }
            else {
                std::cout << std::endl << "Argument " << argv[n] << " does not correspond to any valid arguments. Type -help for details about valid command line arguments." <<std::endl;
		        exit(1);
            }
        }


    }
	else {
		std::cout << std::endl << std::endl << "Not enough arguments. Type -help option in the command line for further explanations." << std::endl;
		exit(1);
	}

    std::cout << "    mode		 = " 	<< mode             << std::endl;
    std::cout << "    threadNum		= " << threadNum        << std::endl;
    std::cout << "    blockNum		= " << blockNum         << std::endl;
    std::cout << "    wuDevice		= " << warm_up_device   << std::endl << std::endl;

    
    //Copying the key file
    unsigned char key[16];
    FILE * keyFile;
    keyFile = fopen(keyfilename,"rb");
    if (keyFile == NULL) {
        perror ("Error opening file");
        exit(1);
    }
    else {
        for(int i=0 ; i<16 ; i+=4) {
            if(fscanf(keyFile, "%x", (unsigned int *)&key[i]) != 1 ) {
                perror ("Error reading keyfile. Make sure the key is hexadecimal words like \"0x01234567 0x89abcdef ...\" .\n");
                exit(1);
            }
        }
    }
    fclose(keyFile);


    // ***Key scheduling***
    uint8 expkey[176];
    ExpandKey (key, expkey);
    hipMemcpyToSymbol(HIP_SYMBOL(const_expkey),  expkey, 176*sizeof(uint8)); //Moving the expanding key to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(const_IK0),  IK0, 256*sizeof(uint32_t));

    // ***Inputdata file to encrypt/decrypt***
    //Checking for the size of the file
    int filesize;
    filesize = fsize(filename);

    //CMS padding to have 16 bytes blocks of data
    uint8_t padElmt;
    int mod16 = filesize%16;

    padElmt = 16 - mod16; // We always add bytes for later padding detection

    //Creating required arrays
    uint8_t *inputData;
    uint8_t *outputData;
    inputData = (uint8_t*)malloc((filesize+padElmt)*sizeof(uint8_t));
    outputData = (uint8_t*)malloc((filesize+padElmt)*sizeof(uint8_t));

    //Opening the file
    FILE * inputFile;
    int result;
    inputFile = fopen(filename,"rb");

    if (inputFile == NULL) {
        perror ("Error opening file");
        exit(1);
    }
    result = fread (inputData, sizeof(uint8_t), filesize, inputFile);
    if(result != filesize) {
        perror("Reading error from the input file");
        exit(1);
    }
    fclose(inputFile);

    //Padding
    for (int i = 0; i < padElmt; i++) {
		inputData[filesize + i] = padElmt;
    }
	filesize += padElmt;
    std::cout << "    Data to treat with padding elements: " << filesize  << " bytes."  << std::endl;

    //Determining grid size if not given
    if(!blockNum) {
        blockNum = 1+filesize/(threadNum*16);
    }
    else {
        if(blockNum*threadNum* 16 < filesize) {
            std::cerr << std::endl <<  std::endl << "BlockNum and ThreadNum don't fit the data file ton encrypt/decrypt. ";
            exit(1);
        }
    }
    std::cout << "    Gridsize in term of block: " << blockNum  << std::endl;

    //Device vectors declarations and allocations
    uint32_t * devInput, * devOutput, * dev_sm_te1, * dev_sm_te2, * dev_sm_te3, * dev_sm_te4;
    uint8_t  * dev_sm_sbox;
    hipMalloc( (void **) &devInput         , filesize*sizeof(uint8_t));
    hipMalloc( (void **) &devOutput        , filesize*sizeof(uint8_t));
    hipMalloc( (void **) &dev_sm_te1       , 256*sizeof(uint32_t));
    hipMalloc( (void **) &dev_sm_te2       , 256*sizeof(uint32_t));
    hipMalloc( (void **) &dev_sm_te3       , 256*sizeof(uint32_t));
    hipMalloc( (void **) &dev_sm_te4       , 256*sizeof(uint32_t));
    hipMalloc( (void **) &dev_sm_sbox      , 256*sizeof(uint8_t));


    //GPU + memory transfers time
    hipEvent_t startHost, stopHost;
	checkCudaErrors(hipEventCreate(&startHost));
	checkCudaErrors(hipEventCreate(&stopHost));

    //To record device time execution
    hipEvent_t startDevice, stopDevice;
	checkCudaErrors(hipEventCreate(&startDevice));
	checkCudaErrors(hipEventCreate(&stopDevice));


    //Copy vectors from host memory to device memory
    if(mode) {
        hipMemcpy(dev_sm_te1       , TBox0         , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te2       , TBox1         , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te3       , TBox2         , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te4       , TBox3         , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_sbox      , sbox          , 256*sizeof(uint8_t ), hipMemcpyHostToDevice);
    }
    else {
        hipMemcpy(dev_sm_te1       , TBoxi0            , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te2       , TBoxi1            , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te3       , TBoxi2            , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te4       , TBoxi3            , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_sbox      , sbox_inv          , 256*sizeof(uint8_t ), hipMemcpyHostToDevice);
    }

    hipMemcpy(devInput, inputData, filesize*sizeof(uint8_t), hipMemcpyHostToDevice);

    //Warm Up
    if(mode) {
        for(int i=0; i < warm_up_device ; i++) { 
            encrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize, dev_sm_te1,
        dev_sm_te2, dev_sm_te3, dev_sm_te4, dev_sm_sbox);
        }
    }
    else {
        for(int i=0; i < warm_up_device ; i++) {
                decrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize, dev_sm_te1,
            dev_sm_te2, dev_sm_te3, dev_sm_te4, dev_sm_sbox);
        }
    }



    #ifdef BENCH_ON
        printf("\nBENCH_ON\n");
        if(mode) {
            checkCudaErrors(hipEventRecord(startHost, NULL));
            for(int j=0; j<1000; j++){ //for benchmarking
                hipMemcpy(devInput, inputData, filesize*sizeof(uint8_t), hipMemcpyHostToDevice);
                //checkCudaErrors(hipEventRecord(startDevice, NULL));
                encrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize, dev_sm_te1,
                dev_sm_te2, dev_sm_te3, dev_sm_te4, dev_sm_sbox);
                //checkCudaErrors(hipEventRecord(stopDevice, NULL));	
                hipMemcpy(outputData, devOutput, filesize*sizeof(uint8_t), hipMemcpyDeviceToHost);	
            }
            checkCudaErrors(hipEventRecord(stopHost, NULL));
        } 
        else {
            checkCudaErrors(hipEventRecord(startHost, NULL));
            for(int j=0; j<1000; j++){ //for benchmarking
                hipMemcpy(devInput, inputData, filesize*sizeof(uint8_t), hipMemcpyHostToDevice);
                //checkCudaErrors(hipEventRecord(startDevice, NULL));						
                decrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize, dev_sm_te1,
                dev_sm_te2, dev_sm_te3, dev_sm_te4, dev_sm_sbox);
                //checkCudaErrors(hipEventRecord(stopDevice, NULL));
                hipMemcpy(outputData, devOutput, filesize*sizeof(uint8_t), hipMemcpyDeviceToHost);
            }
            checkCudaErrors(hipEventRecord(stopHost, NULL));
        }
    #else
        if(mode) {
            checkCudaErrors(hipEventRecord(startHost, NULL));
            hipMemcpy(devInput, inputData, filesize*sizeof(uint8_t), hipMemcpyHostToDevice);
            //checkCudaErrors(hipEventRecord(startDevice, NULL));
            encrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize, dev_sm_te1,
            dev_sm_te2, dev_sm_te3, dev_sm_te4, dev_sm_sbox);
            //checkCudaErrors(hipEventRecord(stopDevice, NULL));		
        }
        else {
            checkCudaErrors(hipEventRecord(startHost, NULL));
            hipMemcpy(devInput, inputData, filesize*sizeof(uint8_t), hipMemcpyHostToDevice);
            //checkCudaErrors(hipEventRecord(startDevice, NULL));						
            decrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize, dev_sm_te1,
            dev_sm_te2, dev_sm_te3, dev_sm_te4, dev_sm_sbox);
            //checkCudaErrors(hipEventRecord(stopDevice, NULL));
        }
        hipMemcpy(outputData, devOutput, filesize*sizeof(uint8_t), hipMemcpyDeviceToHost);
        checkCudaErrors(hipEventRecord(stopHost, NULL));
    #endif
    

    //Make sure to end following events before continuing 
    //checkCudaErrors(hipEventSynchronize(stopDevice));	
    checkCudaErrors(hipEventSynchronize(stopHost));

    //Time calculation
    //float Devmsec   = 0.0f;
    float Hostmsec  = 0.0f;
    double throughput;
    
    /*checkCudaErrors(hipEventElapsedTime(&Devmsec, startDevice, stopDevice));
    throughput = 1.0e-9f*8*filesize/(Devmsec*1.0e-3f);
    printf("\n	GPU processing time: %f (ms)", Devmsec);
    printf("\n	GPU throughput: %f (Gbps)\n", throughput);*/
    
    checkCudaErrors(hipEventElapsedTime(&Hostmsec, startHost, stopHost));
    #ifdef BENCH_ON
        Hostmsec /= 1000;
    #endif
    throughput = 1.0e-9f*8*filesize/(Hostmsec*1.0e-3f);
    printf("\n	Total processing time: %f (ms)", Hostmsec);
    printf("\n	Total throughput: %f (Gbps)\n", throughput);

    //Writing results inside a file
    FILE * outputFile;
    outputFile = fopen("Result/result.dat","wb");

    if (outputFile == NULL) {
        perror ("Error opening file");
        exit(1);
    }
    if(mode){
        result = fwrite (outputData, sizeof(uint8_t), filesize, outputFile);
        if(result != filesize) {
            perror("Writting error to the output file");
            exit(1);
        }
    }
    else {
        padElmt += outputData[filesize-padElmt-1];
        result = fwrite (outputData, sizeof(uint8_t), filesize-padElmt, outputFile);
        if(result != filesize-padElmt) {
            perror("Writting error to the output file");
            exit(1);
        }
    }
    fclose(outputFile);

    //Free host memory
    free(inputData);
    free(outputData);
    // Free device memory
    hipFree(devInput);
    hipFree(devOutput);
    hipFree(dev_sm_te1);
    hipFree(dev_sm_te2);
    hipFree(dev_sm_te3);
    hipFree(dev_sm_te4);
    hipFree(dev_sm_sbox);
    
	return 0;

}
