
/***************************************************************************
 *   Copyright (C) 2020                                                    *
 *                                                                         *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ***************************************************************************/


/**
	@author Broux Paul-Emmanuel <paulemmanuelb@gmail.com>
 */

#define BENCH_ON //Deactivate print messages and introduce warm up and average timing
#define ADV_BTS

#include <stdio.h>
#include <cstdlib>

#include "BtsUtils.h"
#include "transpose.h"

#ifdef ADV_BTS
#include "encryptKernelECB_Advanced.h"
#else
#include "encryptKernelECB.h"
#endif //ADV_BTS

int main(int argc, char * argv[]) {


    ///////////////////////////////////////////////////////////////
    // command line arguments
    ///////////////////////////////////////////////////////////////
    int     warm_up_device      = 0;    // GPU kernel warm up
    int     threadNum           = 64;  // Threads per block. This is a recommanded number.
    int     blockNum            = 0;    // Number of blocks in the grid
    char *  filename;
    char *  keyfilename; 

    std::cout << std::endl << "********************************************************************" ;
    std::cout << std::endl << "******                   AES-128 CUDA                         ******" ;
    std::cout << std::endl << "******                    Bitslicing                          ******" ;
    std::cout << std::endl << "********************************************************************" << std::endl << std::endl; 

    if (argc > 1){
        for( int n=1 ; n<argc ; n=n+2 ) {
            if((strcmp(argv[n],"-wuDevice") == 0) && (n+1<argc)) {
                warm_up_device = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-threadNum") == 0) && (n+1<argc)) {
                threadNum = atoi(argv[n+1]);
                if(threadNum < 0) {
                    printf("\n threadNum must be greater than 0.\n");
                    exit(1);
                }
                #ifdef ADV_BTS
                if(threadNum > 64) {
                    printf("\n threadNum must be lesser than 65.\n");
                    exit(1);
                }
                #endif
            }
            else if((strcmp(argv[n],"-blockNum") == 0) && (n+1<argc)) {
                blockNum = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-filename") == 0) && (n+1<argc)) {
                filename = argv[n+1];
            }
            else if((strcmp(argv[n],"-keyfilename") == 0) && (n+1<argc)) {
                keyfilename = argv[n+1];
            }
            else if((strcmp(argv[n],"-help") == 0)) {
                std::cout << "   This is a AES-128 implementation." << std::endl;
				std::cout << "   \"-options value\" availables are:" << std::endl;
                std::cout << "   -filename, the file path to encrypt or decrypt." << std::endl;
                std::cout << "   -keyfilename, the 128 key file's path to use for encyption or decryption." << std::endl;
                std::cout << "   -threadNum to set the number of threads per block. Default recommended value is 512." << std::endl;
                std::cout << "   -blockNum to set the number of blocks in the grid. Default value is 0 and will create enough blocks taking into account the input file size and the threadNum argument." << std::endl;
                std::cout << "   -wuDevice number of device kernel launch before timing (around 1000 or less is adviced) " << std::endl;
                std::cout << "   The order between options is not important." << std::endl << std::endl;
                exit(0);
            }
            else {
                std::cout << std::endl << "Argument " << argv[n] << " does not correspond to any valid arguments. Type -help for details about valid command line arguments." <<std::endl;
		        exit(1);
            }
        }


    }
	else {
		std::cout << std::endl << std::endl << "Not enough arguments. Type -help option in the command line for further explanations." << std::endl;
		exit(1);
	}

    std::cout << "    threadNum		= " << threadNum        << std::endl;
    std::cout << "    blockNum		= " << blockNum         << std::endl;
    std::cout << "    wuDevice		= " << warm_up_device   << std::endl << std::endl;


    //Copying the key file
    unsigned char key[16];
    FILE * keyFile;
    keyFile = fopen(keyfilename,"rb");
    if (keyFile == NULL) {
        perror ("Error opening file");
        exit(1);
    }
    else {
        for(int i=0 ; i<16 ; i+=4) {
            if(fscanf(keyFile, "%x", (unsigned int *)&key[i]) != 1 ) {
                perror ("Error reading keyfile. Make sure the key is hexadecimal words like \"0x01234567 0x89abcdef ...\" .\n");
                //exit(1);
            }
        }
    }
    fclose(keyFile);

    // ***Key scheduling***
    uint8 expkey[176];
    ExpandKey (key, expkey);

    //Key transposition
    uint32_t transposed_key[1408] = {0};
    transposeKey((uint32_t*)expkey, transposed_key);

    //PRINT KEY AND TRANSPOSED KEY
    printf("\nEncryption key : ");
    for(int i=0; i<16; i++){
        printf("%2x ",key[i]);
    }

    printf("\nRoundKeys\n");
    for(int i=0; i<11; i++){
        printf("Round %2d   ",i);
        print_state_128(&transposed_key[128*i],0);
    }


    //Attach to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(const_expkey),  transposed_key, 1408*sizeof(uint32_t)); //Moving the expanding key to constant memory



    // ***Inputdata file to encrypt/decrypt***
    //Checking for the size of the file
    int filesize;
    filesize = fsize(filename);

    //CMS padding to have 512 bytes blocks of data
    uint32_t padElmt;
/*     uint32_t mod512 = filesize%512;
    padElmt = 512 - mod512; */
    uint32_t mod512 = filesize%16384;
    padElmt = 16384 - mod512;

    filesize += padElmt;

    //Creating required arrays
    uint8_t inputData[filesize] = {0};
    uint8_t outputData[filesize] = {0};

    //Opening the file
    FILE * inputFile;
    int result;
    inputFile = fopen(filename,"rb");

    if (inputFile == NULL) {
        perror ("Error opening file");
        exit(1);
    }
    result = fread (inputData, sizeof(uint8_t), filesize-padElmt, inputFile);
    if(result != filesize-padElmt) {
        perror("Reading error from the input file");
        exit(1);
    }
    fclose(inputFile);

#ifndef BENCH_ON
    //PRINT PLAIN 
    /*printf("\nPlaintext : \n");
    for(int i=0; i<filesize; i++){
        if(i%16==0 && i)
            printf("| ");
        if(i%32==0 && i)
            printf("\n");
        printf("%2x ",inputData[i]);
    }*/
#endif //BENCH_ON

    std::cout << std::endl << "Data to treat with padding elements: " << filesize  << " bytes."  << std::endl;

    //Transposition for bitslicing
    uint32_t *transposition  = (uint32_t*)inputData;
    for(int i=0; i<filesize/512; i++){
        transpose(transposition);
        transposition += 128;
    }

#ifdef BENCH_ON
/*    //PRINT PLAIN TRANSPOSED
    printf("\n Plain through transposed state : \n");
    for(int i=0; i<filesize/512; i++){
        if(i%16==0 && i)
            printf("| ");
        if(i%32==0 && i)
            printf("\n");
        for(int state=0; state<32; state++)
            print_state_128(&transposition[i*128], state);
    }
*/
#endif //BENCH_ON

    //Determining grid size if not given
    if(!blockNum) {
        blockNum = 1+filesize/(threadNum*512);
    }
    std::cout << "Gridsize in term of block: " << blockNum  << std::endl;

    //Device vectors declarations and allocations
    uint32_t * devInput, * devOutput;
    hipMalloc( (void **) &devInput  , filesize*sizeof(uint8_t));
    hipMalloc( (void **) &devOutput , filesize*sizeof(uint8_t));

    //GPU + memory transfers time
    hipEvent_t startHost, stopHost;
	checkCudaErrors(hipEventCreate(&startHost));
	checkCudaErrors(hipEventCreate(&stopHost));

    //To record device time execution
    hipEvent_t startDevice, stopDevice;
	checkCudaErrors(hipEventCreate(&startDevice));
	checkCudaErrors(hipEventCreate(&stopDevice));

    hipMemcpy(devInput, inputData, filesize*sizeof(uint8_t), hipMemcpyHostToDevice);

    //Warm Up
    for(int i=0; i < warm_up_device ; i++) { 
        encrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize);
    }

#ifdef BENCH_ON
    printf("\nBENCH_ON\n");
    checkCudaErrors(hipEventRecord(startHost, NULL));
    for(int j=0; j<1000; j++){ //for benchmarking
        encrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize);
    }
    //checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipEventRecord(stopHost, NULL));
#else
    checkCudaErrors(hipEventRecord(startHost, NULL));
    //hipMemcpy(devInput, inputData, filesize*sizeof(uint8_t), hipMemcpyHostToDevice);
    encrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipEventRecord(stopHost, NULL));
#endif

    checkCudaErrors(hipEventSynchronize(stopHost));
    hipMemcpy(outputData, devOutput, filesize*sizeof(uint8_t), hipMemcpyDeviceToHost);

    //Time spent
    float Hostmsec  = 0.0f;
    double throughput;
    checkCudaErrors(hipEventElapsedTime(&Hostmsec, startHost, stopHost));
#ifdef BENCH_ON
    Hostmsec /= 1000;
#endif
    throughput = 1.0e-9f*8*filesize/(Hostmsec*1.0e-3f);
    printf("\nTotal processing time: %f (ms)", Hostmsec);
    printf("\nTotal throughput: %f (Gbps)\n", throughput);

    //Transposing back
    transposition  = (uint32_t*)outputData;
#ifndef BENCH_ON
    //Print
    /*printf("\n Cipher Text : \n");
    for(int i=0; i<filesize/512; i++){
        if(i%16==0 && i)
            printf("| ");
        if(i%32==0 && i)
            printf("\n");
        for(int state=0; state<32; state++)
            print_state_128(&transposition[i*128], state);
    }*/
#endif //BENCH_ON    

    for(int i=0; i<filesize/512; i++){
        invTranspose(transposition);
        transposition += 128;
    }

#ifndef BENCH_ON
/*
    //PRINT CIPHER 
    printf("\nCiphertext : \n");
    for(int i=0; i<filesize; i++){
        if(i%16==0 && i)
            printf("| ");
        if(i%32==0 && i)
            printf("\n");
        printf("%2x ",outputData[i]);
    }
*/
#endif //BENCH_ON

    //Writing results inside a file
    FILE * outputFile;
    outputFile = fopen("Result/resultBts.dat","wb");

    if (outputFile == NULL) {
        perror ("Error opening file");
        exit(1);
    }
    result = fwrite (outputData, sizeof(uint8_t), filesize-padElmt, outputFile);
    if(result != filesize-padElmt) {
        perror("Writting error to the output file");
        exit(1);
    }
    fclose(outputFile); 

    // Free device memory
    hipFree(devInput);
    hipFree(devOutput);
	return 0;

}
