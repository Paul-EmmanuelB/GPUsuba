#include "hip/hip_runtime.h"

/***************************************************************************
 *   Copyright (C) 2020                                                    *
 *                                                                         *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ***************************************************************************/


/**
	@author Broux Paul-Emmanuel <paulemmanuelb@gmail.com>
 */

#define BENCH_ON //Deactivate print messages and introduce warm up and average timing
#define T uint32_t //Data type to treat in kernels
//#define ADV_BTS

#include <stdio.h>
#include <cstdlib>

#include "BtsUtils.h"
#include "transpose.h"
#include "occupancy.h"

#ifdef ADV_BTS
#include "encryptKernelECB_1.h"
#else
//#include "encryptKernelECB_2.h"
#include "encryptKernelECB_3.h"
//#include "encrypttest.h"
#endif //ADV_BTS

int main(int argc, char * argv[]) {


    ///////////////////////////////////////////////////////////////
    // Command line arguments
    ///////////////////////////////////////////////////////////////
    int     warm_up_device      = 0;    //** GPU kernel warm up
    int     threadNum           = 64;   //** Threads per block. This is a recommanded number.
    int     blockNum            = 16;   //** Number of blocks in the grid
    char *  filename;
    char *  keyfilename; 

    std::cout << std::endl << "********************************************************************" ;
    std::cout << std::endl << "******                   AES-128 CUDA                         ******" ;
    std::cout << std::endl << "******                    Bitslicing                          ******" ;
    std::cout << std::endl << "********************************************************************" << std::endl << std::endl; 

    if (argc > 1){
        for( int n=1 ; n<argc ; n=n+2 ) {
            if((strcmp(argv[n],"-wuDevice") == 0) && (n+1<argc)) {
                warm_up_device = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-threadNum") == 0) && (n+1<argc)) {
                threadNum = atoi(argv[n+1]);
                if(threadNum < 0) {
                    printf("\n threadNum must be greater than 0.\n");
                    exit(1);
                }
            #ifdef ADV_BTS
                if(threadNum > 64) {
                    printf("\n threadNum must be lesser than 65.\n");
                    exit(1);
                }
                if(threadNum%32 != 0) {
                    printf("\n threadNum must be a multiple of warpSize (32).\n");
                    exit(1);
                }
            #endif //ADV_BTS
            }
            else if((strcmp(argv[n],"-blockNum") == 0) && (n+1<argc)) {
                blockNum = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-filename") == 0) && (n+1<argc)) {
                filename = argv[n+1];
            }
            else if((strcmp(argv[n],"-keyfilename") == 0) && (n+1<argc)) {
                keyfilename = argv[n+1];
            }
            else if((strcmp(argv[n],"-help") == 0)) {
                std::cout << "   This is a AES-128 implementation." << std::endl;
				std::cout << "   \"-options value\" availables are:" << std::endl;
                std::cout << "   -filename, the file path to encrypt or decrypt." << std::endl;
                std::cout << "   -keyfilename, the 128 key file's path to use for encyption or decryption." << std::endl;
                std::cout << "   -threadNum to set the number of threads per block. Default recommended value is 64." << std::endl;
                std::cout << "   It is highly recommended to tune that value with multiple of warpSize (32) for better performance." << std::endl;
                std::cout << "   Furthermore, Advanced version need threadNum to be a multiple of warpSize." << std::endl;
                std::cout << "   -blockNum to set the number of blocks in the grid. Default value is 0 and will create enough blocks taking into account the input file size and the threadNum argument." << std::endl;
                std::cout << "   -wuDevice number of device kernel launch before timing (around 1000 or less is adviced) " << std::endl;
                std::cout << "   The order between options is not important." << std::endl << std::endl;
                exit(0);
            }
            else {
                std::cout << std::endl << "Argument " << argv[n] << " does not correspond to any valid arguments. Type -help for details about valid command line arguments." <<std::endl;
		        exit(1);
            }
        }


    }
	else {
		std::cout << std::endl << std::endl << "Not enough arguments. Type -help option in the command line for further explanations." << std::endl;
		exit(1);
	}

    std::cout << "    threadNum		= " << threadNum        << std::endl;
    std::cout << "    blockNum		= " << blockNum         << std::endl;
    std::cout << "    wuDevice		= " << warm_up_device   << std::endl << std::endl;

    // ***Key***
    //* Copying the key file
    unsigned char key[16];
    FILE * keyFile;
    keyFile = fopen(keyfilename,"rb");
    if (keyFile == NULL) {
        perror ("Error opening file");
        exit(1);
    }
    else {
        for(int i=0 ; i<16 ; i+=4) {
            if(fscanf(keyFile, "%x", (unsigned int *)&key[i]) != 1 ) {
                perror ("Error reading keyfile. Make sure the key is hexadecimal words like \"0x01234567 0x89abcdef ...\" .\n");
                exit(1);
            }
        }
    }
    fclose(keyFile);

    //* Key scheduling
    uint8 expkey[176];
    ExpandKey (key, expkey);

    //* Key transposition
    uint32_t transposed_key[1408] = {0};
    transposeKey((uint32_t*)expkey, transposed_key);

    //* Printing key
    printf("\nEncryption key : ");
    for(int i=0; i<16; i++){
        printf("%2x ",key[i]);
    }

    //* Printing round keys
    printf("\nRoundKeys\n");
    for(int i=0; i<11; i++){
        printf("Round %2d   ",i);
        print_state_128(&transposed_key[128*i],0);
    }

    //* Attach to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(const_expkey),  transposed_key, 1408*sizeof(uint32_t));


    // ***Inputdata file to encrypt/decrypt***
    //* Checking for the size of the file
    int filesize;
    filesize = fsize(filename);

    //* CMS padding to have 16KB blocks of data
    uint32_t padElmt;
    uint32_t mod16 = filesize%16384;
    padElmt = 16384 - mod16;
    filesize += padElmt;

    //* Creating required arrays
    uint8_t* inputData;
    uint8_t* outputData;
    inputData = (uint8_t*)malloc(filesize * sizeof(uint8_t));
    outputData = (uint8_t*)malloc(filesize * sizeof(uint8_t));

    for (int i = 0; i < filesize; i++) {
        inputData[i] = 0;
        outputData[i] = 0;
    }

    //* Opening the file
    FILE * inputFile;
    int result;
    inputFile = fopen(filename,"rb");

    if (inputFile == NULL) {
        perror ("Error opening file");
        exit(1);
    }
    result = fread (inputData, sizeof(uint8_t), filesize-padElmt, inputFile);
    if(result != filesize-padElmt) {
        perror("Reading error from the input file");
        exit(1);
    }
    fclose(inputFile);

#ifndef BENCH_ON
    //* Printing plaintext 
    printf("\nPlaintext : \n");
    for(int i=0; i<16; i++){
        printf("%2x ",inputData[i]);
    }
    printf("\n");
#endif //BENCH_ON

    std::cout << std::endl << "Data to treat with padding elements: " << filesize  << " bytes."  << std::endl;

    // ***Transposition for bitslicing***
#ifdef ADV_BTS
    //** Typical bitsliced transposition with 32 states for 128 registers
    uint32_t * transp = (uint32_t*)inputData;
    for(int i=0; i<filesize/512; i++){
        //transpose( (uint32_t*)&inputData[i*128] );
        transpose( transp );
        transp+=128;
    }
#else
    //** Transposition with reorganization within each 1024 states
    //** As such that a warp will read all i-bit position registers at i instruction
    uint32_t * transp = (uint32_t*)inputData;
    for(int i=0; i<filesize/16384; i++){
        //transposeBts( (uint32_t*)&inputData[i*4096] );
        transposeBts( transp );
        transp+=4096;
    }
#endif //ADV_BTS

    //* Printing state from transposed array
#ifndef BENCH_ON
    printf("\nPlain through transposed state : \n");
    #ifdef ADV_BTS
    //** From basic transposed array
    print_state_128((uint32_t*)inputData, 0);
    #else
    //* From reorganized transposed array
    print_state_4096((uint32_t*)inputData, 0);
    #endif //ADV_BTS
#endif //BENCH_ON

    //* Determining grid size if not given
    if(!blockNum) {
        blockNum = 1+filesize/(threadNum*512);
    }
    std::cout << "Gridsize in term of block: " << blockNum  << std::endl;

    // ***Device allocations and transfers***
    //* Device vectors declarations and allocations
    T * devInput, * devOutput;
    hipMalloc( (void **) &devInput  , filesize*sizeof(uint8_t));
    hipMalloc( (void **) &devOutput , filesize*sizeof(uint8_t));

    //* GPU + memory transfers time
    hipEvent_t startHost, stopHost;
	checkCudaErrors(hipEventCreate(&startHost));
	checkCudaErrors(hipEventCreate(&stopHost));

    //* To record device time execution
    hipEvent_t startDevice, stopDevice;
	checkCudaErrors(hipEventCreate(&startDevice));
	checkCudaErrors(hipEventCreate(&stopDevice));

    //* Transferring inputData to the device memory of the GPU 
    hipMemcpy(devInput, inputData, filesize*sizeof(uint8_t), hipMemcpyHostToDevice);


    //  ***Launching kernels***
#ifdef BENCH_ON
    printf("\nBENCH_ON\n");
    //* Warm Up
    for(int i=0; i < warm_up_device ; i++){ 
        encrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize);
    }

    #ifdef ADV_BTS
    //** ADV_BTS kernel with static shared
    checkCudaErrors(hipEventRecord(startHost, NULL));
    for(int j=0; j<100; j++){ //** For benchmarking
        encrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize);
    }
    //checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipEventRecord(stopHost, NULL));
    #else
    //** Kernel with dynamic shared
    checkCudaErrors(hipEventRecord(startHost, NULL));
    for(int j=0; j<100; j++){ //** For benchmarking
        //** Third argument is shared memory space per block. Each warp need 128*sizeof(uint32_t)
        //encrypt_Kernel<<<blockNum,threadNum,4*threadNum*sizeof(uint32_t)>>>(devInput, devOutput, filesize);
		encrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize);
    }
    //checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipEventRecord(stopHost, NULL));
    #endif //ADV_BTS
#else
    #ifdef ADV_BTS
    checkCudaErrors(hipEventRecord(startHost, NULL));
    //hipMemcpy(devInput, inputData, filesize*sizeof(uint8_t), hipMemcpyHostToDevice);
    encrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipEventRecord(stopHost, NULL));
    #else
    checkCudaErrors(hipEventRecord(startHost, NULL));
    //hipMemcpy(devInput, inputData, filesize*sizeof(uint8_t), hipMemcpyHostToDevice);
    //encrypt_Kernel<<<blockNum,threadNum,4*threadNum*sizeof(uint32_t)>>>(devInput, devOutput, filesize);
	encrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipEventRecord(stopHost, NULL));
    #endif //ADV_BTS
#endif

    checkCudaErrors(hipEventSynchronize(stopHost));
    hipMemcpy(outputData, devOutput, filesize*sizeof(uint8_t), hipMemcpyDeviceToHost);

    // ***Time spent***
    float Hostmsec  = 0.0f;
    double throughput;
    checkCudaErrors(hipEventElapsedTime(&Hostmsec, startHost, stopHost));
#ifdef BENCH_ON
    Hostmsec /= 100;
#endif
    throughput = 1.0e-9f*8*filesize/(Hostmsec*1.0e-3f);
    printf("\nTotal processing time: %f (ms)", Hostmsec);
    printf("\nTotal throughput: %f (Gbps)\n", throughput);

    // ***Transposing back***
#ifndef BENCH_ON
    //* Print
    printf("\nCipher Text from transposed output: \n");
    #ifdef ADV_BTS
    print_state_128((uint32_t*)outputData, 0);
    #else
    print_state_4096((uint32_t*)outputData, 0);
    #endif //ADV_BTS
#endif //BENCH_ON    

    #ifdef ADV_BTS
    transp  = (uint32_t*)outputData;
    for(int i=0; i<filesize/512; i++){
        //invTranspose( (uint32_t*)&outputData[128*i] );
        invTranspose( transp );
        transp+=128;
    }
    #else
    transp  = (uint32_t*)outputData;
    for(int i=0; i<filesize/16384; i++){
        //invTransposeBts( (uint32_t*)&outputData[i*4096] );
        invTransposeBts( transp );
        transp+=4096;
    }
    #endif //ADV_BTS

#ifndef BENCH_ON
    //PRINT CIPHER 
    printf("\nCiphertext : \n");
    for(int i=0; i<16; i++){
        printf("%2x ",outputData[i]);
    }
    printf("\n");


    //Writing results inside a file
    FILE * outputFile;
    outputFile = fopen("Result/resultBts.dat","wb");

    if (outputFile == NULL) {
        perror ("Error opening file");
        exit(1);
    }
    result = fwrite (outputData, sizeof(uint8_t), filesize-padElmt, outputFile);
    if(result != filesize-padElmt) {
        perror("Writting error to the output file");
        exit(1);
    }
    fclose(outputFile); 
#endif //BENCH_ON

    //Free Host memory
    free(inputData);
    free(outputData);

    // Free device memory
    hipFree(devInput);
    hipFree(devOutput);
	return 0;

}
