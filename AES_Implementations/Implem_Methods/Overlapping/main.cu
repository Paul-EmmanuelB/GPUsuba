#include "hip/hip_runtime.h"

/***************************************************************************
 *   Copyright (C) 2020                                                    *
 *                                                                         *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ***************************************************************************/


/**
	@author Broux Paul-Emmanuel <paulemmanuelb@gmail.com>
 */

#define BENCH_ON

#include <stdio.h>
#include <cstdlib>

#include "utils.h"
#include "typedef.h"
#include "encryptKernelECB.h"
#include "decryptKernelECB.h"
#include "sboxE.h"
#include "sboxD.h"


int main(int argc, char * argv[]) {


    ///////////////////////////////////////////////////////////////
    // command line arguments
    ///////////////////////////////////////////////////////////////
    int     warm_up_device      = 0;    // GPU kernel warm up
    int     threadNum           = 512;  // Threads per block. This is a recommanded number.
    int     blockNum            = 0;    // Number of blocks in the grid
    int     streamNum           = 2;    // Number of streams to create for overlapping 
    int     mode                = 1;    // Encryption mode, 1 to encrypt or 0 to decrypt.
    char *  filename;
    char *  keyfilename; 

    std::cout << std::endl << "********************************************************************" ;
    std::cout << std::endl << "******                   AES-128 CUDA                         ******" ;
    std::cout << std::endl << "******                   Overlapping                          ******" ;
    std::cout << std::endl << "********************************************************************" << std::endl << std::endl; 

    if (argc > 1){
        for( int n=1 ; n<argc ; n=n+2 ) {
            if((strcmp(argv[n],"-wuDevice") == 0) && (n+1<argc)) {
                warm_up_device = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-threadNum") == 0) && (n+1<argc)) {
                threadNum = atoi(argv[n+1]);
                if(threadNum ==0) {
                    printf("\n threadNum must be a non-null value.\n");
                    exit(1);
                }
            }
            else if((strcmp(argv[n],"-blockNum") == 0) && (n+1<argc)) {
                blockNum = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-streamNum") == 0) && (n+1<argc)) {
                streamNum = atoi(argv[n+1]);
                if(streamNum ==0) {
                    printf("\n streamNum must be a non-null value.\n");
                    exit(1);
                }
                if(streamNum >= 16) {
                    printf("\n streamNum can't exceed 15.\n");
                    exit(1);
                }
            }
            else if((strcmp(argv[n],"-mode") == 0) && (n+1<argc)) {
                mode = atoi(argv[n+1]);
            }
            else if((strcmp(argv[n],"-filename") == 0) && (n+1<argc)) {
                filename = argv[n+1];
            }
            else if((strcmp(argv[n],"-keyfilename") == 0) && (n+1<argc)) {
                keyfilename = argv[n+1];
            }
            else if((strcmp(argv[n],"-help") == 0)) {
                std::cout << "   This is a AES-128 implementation." << std::endl;
				std::cout << "   \"-options value\" availables are:" << std::endl;
                std::cout << "   -mode, 1 to encrypt and 0 to decrypt. Default value is 1." << std::endl;
                std::cout << "   -filename, the file path to encrypt or decrypt." << std::endl;
                std::cout << "   -keyfilename, the 128 key file's path to use for encyption or decryption." << std::endl;
                std::cout << "   -threadNum to set the number of threads per block. Default recommended value is 512." << std::endl;
                std::cout << "   -blockNum to set the number of blocks in the grid. Default value is 0 and will create" << std::endl << "   enough blocks taking into account the input file size and the threadNum argument." << std::endl;
                std::cout << "   -streamNum will set the number of streams and the number of chunk created from the plaintext. Default value is 2 and max is 15." << std::endl;
                std::cout << "   -wuDevice number of device kernel launch before timing (around 1000 or less is adviced) " << std::endl;
                std::cout << "   The order between options is not important." << std::endl << std::endl;
                exit(0);
            }
            else {
                std::cout << std::endl << "Argument " << argv[n] << " does not correspond to any valid arguments. Type -help for details about valid command line arguments." <<std::endl;
		        exit(1);
            }
        }


    }
	else {
		std::cout << std::endl << std::endl << "Not enough arguments. Type -help option in the command line for further explanations." << std::endl;
		exit(1);
	}

    std::cout << "    mode		 = " 	<< mode             << std::endl;
    std::cout << "    threadNum		= " << threadNum        << std::endl;
    std::cout << "    blockNum		= " << blockNum         << std::endl;
    std::cout << "    streamNum		= " << streamNum        << std::endl;
    std::cout << "    wuDevice		= " << warm_up_device   << std::endl << std::endl;

    
    //Copying the key file
    unsigned char key[16];
    FILE * keyFile;
    keyFile = fopen(keyfilename,"rb");
    if (keyFile == NULL) {
        perror ("Error opening file");
        exit(1);
    }
    else {
        for(int i=0 ; i<16 ; i+=4) {
            if(fscanf(keyFile, "%x", (unsigned int *)&key[i]) != 1 ) {
                perror ("Error reading keyfile. Make sure the key is hexadecimal words like \"0x01234567 0x89abcdef ...\" .\n");
                exit(1);
            }
        }
    }
    fclose(keyFile);


    // ***Key scheduling***
    uint8 expkey[176];
    ExpandKey (key, expkey);
    hipMemcpyToSymbol(HIP_SYMBOL(const_expkey),  expkey, 176*sizeof(uint8)); //Moving the expanding key to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(const_IK0),  IK0, 256*sizeof(uint32_t));

    // ***Inputdata file to encrypt/decrypt***
    //Checking for the size of the file
    int filesize;
    filesize = fsize(filename);

    //CMS padding to have 16 bytes blocks of data
    uint8 padElmt = 0;
    uint8 streamPad = 0; 
    int mod16 = filesize%16;

    if(mode){
        padElmt = 16 - mod16; // We always add bytes for later padding detection
        mod16 = ((filesize+padElmt)/streamNum)%16; //padding for making each futur chunk multiple of 16
        streamPad = streamNum*(16 - mod16);
    }

    //Creating required arrays with page-locked memory
    uint8 *hostInData;
    checkCudaErrors(hipHostAlloc((void**)&hostInData, (filesize+padElmt+streamPad)*sizeof(uint8), hipHostMallocDefault));
    
    //Opening the file
    FILE * inputFile;
    int result;
    inputFile = fopen(filename,"rb");

    if (inputFile == NULL) {
        perror ("Error opening file");
        exit(1);
    }
    result = fread (hostInData, sizeof(uint8), filesize, inputFile);
    if(result != filesize) {
        perror("Reading error from the input file");
        exit(1);
    }
    fclose(inputFile);

    //Padding
    for (int i = 0; i < padElmt; i++) {
		hostInData[filesize + i] = padElmt;
    }
    filesize += padElmt;
    for (int i = 0; i < streamPad; i++) {
		hostInData[filesize + i] = streamPad;
    }
    filesize += streamPad;
    std::cout << "    Data to treat with padding elements: " << filesize  << " bytes."  << std::endl;

    //Determining grid size if not given
    int size = filesize/streamNum; // To treat by each kernel launch
    if(!blockNum) {
        blockNum = 1+size/(threadNum*16);
    }
    else {
        if(blockNum*threadNum* 16 < size) {
            std::cerr << std::endl <<  std::endl << "BlockNum and ThreadNum don't fit the data file ton encrypt/decrypt. ";
            exit(1);
        }
    }
    std::cout << "    Gridsize in term of block: " << blockNum  << std::endl;

    //Streams creation
    hipStream_t stream[streamNum];
    for (int i = 0; i < streamNum; ++i)
        hipStreamCreate(&stream[i]);

    //Device vectors declarations and allocations
    uint32_t * devInput, * devOutput, * dev_sm_te1, * dev_sm_te2, * dev_sm_te3, * dev_sm_te4;
    uint8  * dev_sm_sbox;
    hipMalloc( (void **) &devInput         , filesize*sizeof(uint8));
    hipMalloc( (void **) &devOutput        , filesize*sizeof(uint8));
    hipMalloc( (void **) &dev_sm_te1       , 256*sizeof(uint32_t));
    hipMalloc( (void **) &dev_sm_te2       , 256*sizeof(uint32_t));
    hipMalloc( (void **) &dev_sm_te3       , 256*sizeof(uint32_t));
    hipMalloc( (void **) &dev_sm_te4       , 256*sizeof(uint32_t));
    hipMalloc( (void **) &dev_sm_sbox      , 256*sizeof(uint8));


    //GPU + memory transfers time
    hipEvent_t startHost, stopHost, delayHost;
	checkCudaErrors(hipEventCreate(&startHost));
	checkCudaErrors(hipEventCreate(&stopHost));
    checkCudaErrors(hipEventCreate(&delayHost));

    //To record device time execution
    hipEvent_t startDevice, stopDevice;
	checkCudaErrors(hipEventCreate(&startDevice));
	checkCudaErrors(hipEventCreate(&stopDevice));

    
    //Copy vectors from host memory to device memory
    if(mode) {
        hipMemcpy(dev_sm_te1       , TBox0         , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te2       , TBox1         , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te3       , TBox2         , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te4       , TBox3         , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_sbox      , sbox          , 256*sizeof(uint8 ), hipMemcpyHostToDevice);
    }
    else {
        hipMemcpy(dev_sm_te1       , TBoxi0            , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te2       , TBoxi1            , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te3       , TBoxi2            , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_te4       , TBoxi3            , 256*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(dev_sm_sbox      , sbox_inv          , 256*sizeof(uint8 ), hipMemcpyHostToDevice);
    }

    uint32_t *hostData = (uint32_t*)hostInData;
    int word = size/4;

    //Warm Up
    hipMemcpy(devInput, hostInData, filesize*sizeof(uint8_t), hipMemcpyHostToDevice);
    if(mode) {
        for(int i=0; i < warm_up_device ; i++) { 
            encrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize, dev_sm_te1,
        dev_sm_te2, dev_sm_te3, dev_sm_te4, dev_sm_sbox);
        }
    }
    else {
        for(int i=0; i < warm_up_device ; i++) {
                decrypt_Kernel<<<blockNum,threadNum>>>(devInput, devOutput, filesize, dev_sm_te1,
            dev_sm_te2, dev_sm_te3, dev_sm_te4, dev_sm_sbox);
        }
    }


    #ifdef BENCH_ON
        if(mode) {
            printf("\nBENCH_ON\n");
            checkCudaErrors(hipEventRecord(startDevice, NULL));
            for(int j=0; j<1000; j++){
                for(int i=0; i < streamNum ; ++i) {
                    hipMemcpyAsync(devInput+i*word, hostData+i*word, size, hipMemcpyHostToDevice, stream[i]);
                    encrypt_Kernel<<<blockNum,threadNum, 0, stream[i]>>>(devInput+i*word, devOutput+i*word, size, dev_sm_te1,
                    dev_sm_te2, dev_sm_te3, dev_sm_te4, dev_sm_sbox);
                    hipMemcpyAsync(hostData+i*word,  devOutput+i*word, size, hipMemcpyDeviceToHost, stream[i]);
                }
            }
            checkCudaErrors(hipEventRecord(stopDevice, NULL));		
        }
        else {
            checkCudaErrors(hipEventRecord(startDevice, NULL));
            for(int j=0; j<1000; j++){					
                for(int i=0; i < streamNum ; ++i) {
                    hipMemcpyAsync(devInput+i*word, hostData+i*word, size, hipMemcpyHostToDevice, stream[i]);
                    decrypt_Kernel<<<blockNum,threadNum, 0, stream[i]>>>(devInput+i*word, devOutput+i*word, size, dev_sm_te1,
                    dev_sm_te2, dev_sm_te3, dev_sm_te4, dev_sm_sbox);
                    hipMemcpyAsync(hostData+i*word,  devOutput+i*word, size, hipMemcpyDeviceToHost, stream[i]);
                }
            }
            checkCudaErrors(hipEventRecord(stopDevice, NULL));
        }
    #else
        if(mode) {
            checkCudaErrors(hipEventRecord(startDevice, NULL));
            for(int i=0; i < streamNum ; ++i) {
                hipMemcpyAsync(devInput+i*word, hostData+i*word, size, hipMemcpyHostToDevice, stream[i]);
                encrypt_Kernel<<<blockNum,threadNum, 0, stream[i]>>>(devInput+i*word, devOutput+i*word, size, dev_sm_te1,
                dev_sm_te2, dev_sm_te3, dev_sm_te4, dev_sm_sbox);
                hipMemcpyAsync(hostData+i*word,  devOutput+i*word, size, hipMemcpyDeviceToHost, stream[i]);
            }
            checkCudaErrors(hipEventRecord(stopDevice, NULL));		
        }
        else {
            checkCudaErrors(hipEventRecord(startDevice, NULL));						
            for(int i=0; i < streamNum ; ++i) {
                hipMemcpyAsync(devInput+i*word, hostData+i*word, size, hipMemcpyHostToDevice, stream[i]);
                decrypt_Kernel<<<blockNum,threadNum, 0, stream[i]>>>(devInput+i*word, devOutput+i*word, size, dev_sm_te1,
                dev_sm_te2, dev_sm_te3, dev_sm_te4, dev_sm_sbox);
                hipMemcpyAsync(hostData+i*word,  devOutput+i*word, size, hipMemcpyDeviceToHost, stream[i]);
            }
            checkCudaErrors(hipEventRecord(stopDevice, NULL));
        }
    #endif
    
    checkCudaErrors(hipEventSynchronize(stopDevice));	
    checkCudaErrors(hipDeviceSynchronize());

    //Time calculation
    float Devmsec   = 0.0f;
    
    checkCudaErrors(hipEventElapsedTime(&Devmsec, startDevice, stopDevice));
    #ifdef BENCH_ON
        Devmsec/= 1000;
    #endif
    double throughput = 1.0e-9f*8*filesize/(Devmsec*1.0e-3f);
    printf("\n	GPU processing time: %f (ms)", Devmsec);
    printf("\n	GPU throughput: %f (Gbps)\n", throughput);

    //Writing results inside a file
    FILE * outputFile;
    outputFile = fopen("Result/result.dat","wb");

    if (outputFile == NULL) {
        perror ("Error opening file");
        exit(1);
    }
    if(mode){
        result = fwrite (hostInData, sizeof(uint8), filesize, outputFile);
        if(result != filesize) {
            perror("Writting error to the output file");
            exit(1);
        }
    }
    else {
        uint8 padTotal = hostInData[filesize - 1];
        padTotal += hostInData[filesize - padTotal - 1];
        result = fwrite (hostInData, sizeof(uint8), filesize-padTotal, outputFile);
        if(result != filesize-padTotal) {
            perror("Writting error to the output file");
            exit(1);
        }
    }
    fclose(outputFile);

    //free streams
    for (int i = 0; i < streamNum; ++i)
        hipStreamDestroy(stream[i]);

    // Free device memory 
    checkCudaErrors( hipHostFree(hostInData) );  
    hipFree(devInput);
    hipFree(devOutput);
    hipFree(dev_sm_te1);
    hipFree(dev_sm_te2);
    hipFree(dev_sm_te3);
    hipFree(dev_sm_te4);
    hipFree(dev_sm_sbox);
    
	return 0;
}
